#include "hip/hip_runtime.h"
/*
Author: Cao Thanh Tung
Date: 15/03/2011

File Name: cudaReconstruction.cu

This file include all CUDA code to perform the reconstruction step

===============================================================================

Copyright (c) 2011, School of Computing, National University of Singapore. 
All rights reserved.

Project homepage: http://www.comp.nus.edu.sg/~tants/cdt.html

If you use GPU-DT and you like it or have comments on its usefulness etc., we 
would love to hear from you at <tants@comp.nus.edu.sg>. You may share with us
your experience and any possibilities that we may improve the work/code.

===============================================================================

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution. 

Neither the name of the National University of University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission from the National University of Singapore. 

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES 
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.

*/

#pragma warning(disable: 4311 4312)

#include <hip/device_functions.h>
#include <stdio.h>
#include <string.h>
#include "gpudt.h"
#include "cudaDecl.h"
#include "common.h"

#include "cudaCCW.h"

/***********************************************************
 * Declarations
 ***********************************************************/
#define WBLOCK                256        
#define SBLOCK                128
#define MBLOCK                32

/**************************************************************
 * Exported methods
 **************************************************************/
extern "C" void cudaReconstruction();

/**************************************************************
 * Definitions
 **************************************************************/
#define MAX(a, b)            (((a) > (b)) ? (a) : (b))
#define SHIFT                5

// Buffer to be used when constructing vertex array in CPU
#define BUFFER_ITEM        910
#define BUFFER_SIZE        BUFFER_ITEM * 9        // Buffer 32K

// Decode an oriented triangle. 
// An oriented triangle consists of 32 bits. 
// - 30 highest bits represent the triangle index, 
// - 2 lowest bits represent the orientation (the starting vertex, 0, 1 or 2)
#define decode_tri(x)    ((x) >> 2)
#define decode_ori(x)    ((x) & 3)

__constant__ int minus1mod3[4] = {2, 0, 1, -1};
__constant__ int plus1mod3[4] = {1, 2, 0, -1};

// Mark those shifted sites (either can or cannot)
// Shared with the Shifting stage. 
int *cactive;                
/* cactive is used in the shifting stage. 
 * The code used in cactive is as follow: 
 *         0        : Missing sites
 *        -1        : Boundary
 *        -2, 2    : Marked during the reconstruction stage
 *        -x, x   : Different steps in the shifting detection algorithm. 
 *      +inf    : Unprocessed
 *
 * Any value > 0 indicate that that vertex can be shifted safely. 
 */

/************************************************************
 * Variables and functions shared with the main module
 ************************************************************/
extern int *voronoiPtr;        // Discrete voronoi diagram
extern short2 *patternPtr;        // Voronoi vertex patterns

extern int nTris, nVerts, nPoints;    
extern int *ctriangles;        
extern int *cvertarr;        
extern int *tvertices; 
extern REAL2 *cvertices;    
extern REAL2 *covertices;    
extern PGPUDTPARAMS gpudtParams;    
extern int *cactive;                

extern int sizeTexture; 
extern REAL scale, shiftX, shiftY; 

extern int *boundary;
extern gpudtVertex *gpudtVertices; 
extern gpudtTriangle *gpudtTriangles;

extern int gpudtFixConvexHull(int *additionalTriangles, int fboWidth, int *boundary);

/*********************************************************************************
 * Count the number of triangle generated for each row of the texture. 
 * Used to calculate the offset to which each thread processing a texture row 
 * will insert the generated triangles.
 * Also, collect the boundary pixels of the texture to be used in the next CPU step
 *********************************************************************************/
__global__ void kernelCountRow(int *voronoiPtr, short2 *patternPtr, int *count, int width, int min, int max, int *cboundary) {
    // Get the row we are working on
    int x = blockIdx.x * blockDim.x + threadIdx.x; 

    // Collect the boundary (up, left, down, right)
    if (x > 0 && x <= max) {
        cboundary[width * 0 + x] = voronoiPtr[min * width + x]; 
        cboundary[width * 1 + x] = voronoiPtr[x * width + min]; 
        cboundary[width * 2 + x] = voronoiPtr[max * width + x]; 
        cboundary[width * 3 + x] = voronoiPtr[x * width + max]; 
    }

    // Actual counting
    if (x < min || x >= max)
        return ;  

    int xwidth = x * width; 
    int result = 0;
    short2 t = patternPtr[xwidth + min]; 
    
    // Keep jumping and counting
    while (t.y > 0 && t.y < max) {
        result += 1 + (t.x >> 2); 
        t = patternPtr[xwidth + t.y + 1];
    }

    count[x] = result; 
}

/*********************************************************************************
 * Prefix sum on the counted value to calculate the offset
 *********************************************************************************/
void cudaPrefixSum(int *cpuCount, int min, int max) {
    cpuCount[min-1] = 0; 
    for (int i = min; i < max; i++)
        cpuCount[i] += cpuCount[i-1]; 
}

/*********************************************************************************
 * Generate triangles from the Voronoi vertices and insert them into the triangle list.
 *********************************************************************************/
__global__ void kernelGenerateTriangles(int *voronoiPtr, short2 *patternPtr, int3 *ctriangles, 
										int *offset, int width, int min, int max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; 

    if (x < min || x >= max)
        return ;  

    int xwidth = x * width; 
    short2 pattern = patternPtr[xwidth + min];
    int i0, i1, i2, i3;
    int3 *pT = &ctriangles[offset[x-1] * 3 + 1]; 

    // Jump through all voronoi vertices in a texture row
    while (pattern.y > 0 && pattern.y < max) {
        i0 = voronoiPtr[xwidth + pattern.y]; 
        i1 = voronoiPtr[xwidth + pattern.y + 1]; 
        i2 = voronoiPtr[xwidth + width + pattern.y + 1];
        i3 = voronoiPtr[xwidth + width + pattern.y]; 

        if (pattern.x == 0) *pT = make_int3(i3, i1, i2); 
        if (pattern.x == 1) *pT = make_int3(i0, i2, i3); 
        if (pattern.x == 2) *pT = make_int3(i1, i3, i0); 
        if (pattern.x == 3) *pT = make_int3(i2, i0, i1); 
        if (pattern.x == 4) {
            // Generate 2 triangles. 
            // Since the hole is convex, no need to do CCW test
            *pT = make_int3(i2, i0, i1); pT += 3; 
            *pT = make_int3(i3, i0, i2); 
        }
        
        pattern = patternPtr[xwidth + pattern.y + 1]; 
        pT += 3; 
    }
}

/************************************************************
 * Scale back the point set
 ************************************************************/
__global__ void kernelScaleBack(REAL2 *cvertices, REAL scale, REAL shiftX, REAL shiftY, int nPoints) {
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x; 

    if (x >= nPoints)
        return ; 

    REAL2 t = cvertices[x]; 

    t.x = shiftX + t.x * scale; 
    t.y = shiftY + t.y * scale; 

    cvertices[x] = t; 
}

/*********************************************************************************
 * Map all sites to its ID, including missing sites
 *********************************************************************************/
__global__ void kernelMapToId(int *voronoiPtr, int nVerts, REAL2 *cvertices, 
                              int *tvertices, int width) {
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x; 

    if (x >= nVerts)
        return ; 

    REAL2 t = cvertices[x]; 
    tvertices[x] = voronoiPtr[(int(t.y) + 1) * width + (int(t.x) + 1)];
}

/*********************************************************************************
 * Find 3 neighbours sharing one edge with each triangle.
 *********************************************************************************/
__global__ void kernelFindNextTriangles(int *ctriangles, int *cvertarr, int nTris) {
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris) 
        return ; 

    int p0, p1, p2;
    int nCounter, pNextTri, pTri, pOri, pTri9; 
    int x9 = x * 9;

	p0 = ctriangles[x9+4]; 
    p1 = ctriangles[x9+5]; 
    p2 = ctriangles[x9+3]; 
    nCounter = 0; 

	// orientation 0
	// Travel through the list of triangles sharing vertex 0 with this triangle.
	// In this list we can find at most two triangles sharing edge (p0, p1) and 
	// (p2, p0) with our triangle. 
	if ( p1 < p0 || p0 < p2 ) 
	{
		pNextTri = cvertarr[p0];  //ctriangles[x9+6];

		while (pNextTri >= 0 && nCounter < 2) {
			pTri = decode_tri(pNextTri); 
			pOri = decode_ori(pNextTri); 
			pTri9 = pTri * 9; 

			if (p2 == ctriangles[pTri9 + 3 + minus1mod3[pOri]]) {    // NextDest
				ctriangles[x9 + 2] = pNextTri; 
				ctriangles[pTri9 + pOri] = (x << 2) | 2;  
				nCounter++; 
			}

			if (p1 == ctriangles[pTri9 + 3 + pOri]) {    // NextApex
				ctriangles[x9 + 0] = (pTri << 2) | minus1mod3[pOri];  
				ctriangles[pTri9 + minus1mod3[pOri]] = (x << 2);  
				nCounter++; 
			}

			pNextTri = ctriangles[pTri9 + 6 + pOri]; 
		}
	}

    // orientation 1
    // Find the triangle with edge (p1, p2)
	if ( p2 < p1 ) 
	{
		pNextTri = cvertarr[p1]; //ctriangles[x9+7]; 

		while (pNextTri >= 0) {
			pTri = decode_tri(pNextTri); 
			pOri = decode_ori(pNextTri); 
			pTri9 = pTri * 9; 

			if (p2 == ctriangles[pTri9 + 3 + pOri]) {    // NextApex
				ctriangles[x9 + 1] = (pTri << 2) | minus1mod3[pOri]; 
				ctriangles[pTri9 + minus1mod3[pOri]] = (x << 2) | 1;  
				break ; 
			}

			pNextTri = ctriangles[pTri9 + 6 + pOri]; 
		}
	}
}

__global__ void kernelAppendTri(int3* dest, int3* source, int noTris)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; 

	if (x >= noTris) 
		return; 

	dest[x * 3 + 1] = source[x]; 
}

__global__ void kernelSetVertArray(int *ctriangles, int nTris, int *cvertarr) 
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;	
    if (x >= nTris)
        return ; 

    int v0 = ctriangles[x * 9 + 4];
    int v1 = ctriangles[x * 9 + 5];
    int v2 = ctriangles[x * 9 + 3];

    ctriangles[x * 9 + 6] = atomicExch(&cvertarr[v0], (x << 2)); 
    ctriangles[x * 9 + 7] = atomicExch(&cvertarr[v1], (x << 2) | 1); 
    ctriangles[x * 9 + 8] = atomicExch(&cvertarr[v2], (x << 2) | 2); 
}

__global__ void kernelClearNeighbors(int3 * ctriangles, int nTris)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if (x >= nTris) 
		return; 

	ctriangles[x * 3] = make_int3(-1, -1, -1); 
}

//////////////////////////////////////////////////////////////

void cudaReconstruction() {
    int *count,                // Number of triangle generated in one texture row
        *cboundary;            // Boundary pixels of the texture

    dim3 grid, block; 

    /****************************************************************************************
     * Initialization
     ****************************************************************************************/
    int texSize    = gpudtParams->fboSize; 
    int min        = 1; 
    int max        = texSize - 2;    // Assume width = height
    
    // EXACT test constants
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(constData), hostConst, 13 * sizeof(REAL)) ); 

    /****************************************************************************************
     * CUDA: Count the number of Voronoi Vertices in each row of the texture
     ****************************************************************************************/
    cutilSafeCall( hipMalloc((void **) &count, texSize * sizeof(int)) ); 
    cutilSafeCall( hipMalloc((void **) &cboundary, texSize * 4 * sizeof(int)) ); 

    int *cpuCount = (int *) malloc(texSize * sizeof(int)); 
    int *boundary = (int *) malloc(texSize * 4 * sizeof(int)); 

    block = dim3(MBLOCK);    
    grid = dim3(texSize / block.x);
    kernelCountRow<<< grid, block >>>(voronoiPtr, patternPtr, count, texSize, min, max, cboundary);
    cutilCheckError(); 

    cutilSafeCall( hipMemcpy(boundary, cboundary, texSize * 4 * sizeof(int), hipMemcpyDeviceToHost) ); 
    cutilSafeCall( hipMemcpy(cpuCount, count, texSize * sizeof(int), hipMemcpyDeviceToHost) ); 

    /****************************************************************************************
     * CPU: Prefix sum
     ****************************************************************************************/

    cudaPrefixSum(cpuCount, min, max); 
    cutilSafeCall( hipMemcpy(count, cpuCount, texSize * sizeof(int), hipMemcpyHostToDevice) );
    nTris = cpuCount[max-1];

    free(cpuCount); 

    /****************************************************************************************
     * CUDA: Generate triangles
     ****************************************************************************************/
    // We use a very small block size here because there are 
    // very few texture rows, we want to fully utilize the multiprocessors.
    block = dim3(MBLOCK);
    grid = dim3(texSize / block.x);
    kernelGenerateTriangles<<< grid, block >>>(voronoiPtr, patternPtr, (int3 *) ctriangles, 
		count, texSize, min, max);
    cutilCheckError(); 

	block = dim3(WBLOCK);    
    grid = dim3(STRIPE, nPoints / (block.x * STRIPE) + 1); 
    kernelMapToId<<< grid, block >>>(voronoiPtr, nPoints, cvertices, tvertices, texSize); 
    cutilCheckError(); 

    // Fix the convex hull right here. 
    int boundaryTris = gpudtFixConvexHull((int *)gpudtTriangles, texSize, boundary);

	int *convexHullTri; 
    cutilSafeCall( hipMalloc(&convexHullTri, boundaryTris * 3 * sizeof(int)) ); 
	cutilSafeCall( hipMemcpy(convexHullTri, gpudtTriangles, boundaryTris * 3 * sizeof(int), hipMemcpyHostToDevice) ); 

	grid = dim3(boundaryTris / block.x + 1); 

	kernelAppendTri<<< grid, block >>>(((int3 *) ctriangles) + nTris * 3, (int3 *) convexHullTri, boundaryTris); 
    cutilCheckError(); 

    nTris += boundaryTris; 

    free(boundary); 
	cutilSafeCall( hipFree(convexHullTri) ); 
    cutilSafeCall( hipFree(cboundary) ); 
    cutilSafeCall( hipFree(count) ); 

    // Scale the point set back
    block = dim3(WBLOCK); 
    grid = dim3(STRIPE, nPoints / (STRIPE * block.x) + 1); 
    kernelScaleBack<<< grid, block >>>(cvertices, scale, shiftX, shiftY, nPoints); 
    cutilCheckError(); 

	// Calculate the vertex array
	cutilSafeCall( hipMalloc((void **) &cvertarr, nVerts * sizeof(int)) ); 
    block = dim3(128);
    grid = dim3(STRIPE, nTris/(STRIPE * block.x) + 1);

    cutilSafeCall( hipMemset(cvertarr, 255, nVerts * sizeof(int)) );
    kernelSetVertArray<<< grid, block >>>(ctriangles, nTris, cvertarr); //fix vertex array 
    cutilCheckError(); 

	/****************************************************************************************
     * Find next triangles
     ****************************************************************************************/
	// First, fill the next triangles links with -1
    block = dim3(WBLOCK); 
    grid = dim3(STRIPE, nTris / (STRIPE * block.x) + 1); 

	kernelClearNeighbors<<< grid, block >>>((int3 *) ctriangles, nTris); 
    cutilCheckError(); 

    kernelFindNextTriangles<<< grid, block >>>(ctriangles, cvertarr, nTris); 
    cutilCheckError(); 

    /****************************************************************************************
     * Done!!!
     ****************************************************************************************/
}
