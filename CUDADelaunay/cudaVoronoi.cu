#include "hip/hip_runtime.h"
/*
Author: Cao Thanh Tung, Rong Guodong, Stephanus
Date: 15/03/2011

File Name: cudaVoronoi.cu

This file include all CUDA code to perform Voronoi Diagram computation, 
remove islands, etc. 

===============================================================================

Copyright (c) 2011, School of Computing, National University of Singapore. 
All rights reserved.

Project homepage: http://www.comp.nus.edu.sg/~tants/cdt.html

If you use GPU-DT and you like it or have comments on its usefulness etc., we 
would love to hear from you at <tants@comp.nus.edu.sg>. You may share with us
your experience and any possibilities that we may improve the work/code.

===============================================================================

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution. 

Neither the name of the National University of University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission from the National University of Singapore. 

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES 
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.

*/

#pragma warning(disable: 4311 4312)
#include <hip/device_functions.h>
#include <stdio.h>
#include <string.h>
#include "cudaDecl.h"
#include "common.h"
#include<iostream>   
#include<vector>   
#include<queue>   
using namespace std;

extern "C" void cudaDiscreteVoronoiDiagram();
extern "C" void pba2DCompute(int phase1Band, int phase2Band, int phase3Band);

#define MAX_INT        2147483647
#define MIN_SHORT    -32768
#define MAX_SHORT    32767

#define BLOCKX        16
#define BLOCKY        8
#define WBLOCK        BLOCKX * BLOCKY

// Parameters for PBA
int phase1Band = 16;
int phase2Band = 16;
int phase3Band = 16;

 

#define ROUND(x, y)    (((x) - 1) / (y) + 1)

/****** Global Variables *******/
extern PGPUDTPARAMS gpudtParams;        // Input parameters

int iCurrentBuffer; 
int texSize; 
int log2Width; 

texture<short2, 1, hipReadModeElementType> texColor; 
texture<short2, 1, hipReadModeElementType> texPattern; 
texture<int, 1, hipReadModeElementType> texInt; 

// Shared with main module
extern short2 **pingpongColor, **pingpongPattern;   
extern int *voronoiPtr; 
extern short2 *patternPtr; 
extern REAL2 *cvertices;        // Original coordinates of the sites    
extern int sizeTexture; 
extern int *cflag; 
extern short *fordraw2;
/********* Kernels ********/
#include "kernelVoronoi.h"

/****** Helper modules ******/

#define SCALEX(x)    (((x) - minX) * texWidthNeg2 / rangeX)
#define SCALEY(y)    (((y) - minY) * texHeightNeg2 / rangeY) 

struct Node{ 
	short2 island, seed;   
	Node(short2 a, short2 b)
	{       
		island = a;
	    seed = b;
	}
}; 
bool operator<( Node a, Node b )
{    
	int x1 = a.island.x;
	int y1 = a.island.y;
	int x2 = a.seed.x;
	int y2 = a.seed.y;
	int dist1 = (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2);
	x1 = b.island.x;
	y1 = b.island.y;
	x2 = b.seed.x;
	y2 = b.seed.y;
	int dist2 = (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2);
	return dist1>dist2;
} 

void colorIsland(short2 * mapOfIsland, short2 *output, int islands, int texSize)
{
	priority_queue<Node> q;	
    short2 island, seed;	
    int min = 1, max = texSize - 1;	
	for(int t=0; t<islands; t++)
		if (output[t * 2 + 1].x == texSize)
		{		 
			int x,y;
			island = output[t * 2];			
			//check its 8 neighbors
			for (x = island.x-1; x <= island.x+1; x++)
				for (y = island.y-1; y <= island.y+1; y++)
					if (x >= min && x < max && y >= min && y < max)
					{							
						seed = mapOfIsland[texSize * y + x];							
						if (seed.x != texSize)
						{
							int dist1 = (x - seed.x) * (x - seed.x) + (y - seed.y) * (y - seed.y); 
							int dist2 = (island.x - seed.x) * (island.x - seed.x) + 
								(island.y - seed.y) * (island.y - seed.y); 

							if (dist1 <= dist2)							
								q.push(Node(island, seed)); 
						}
					}
		}

	while(!q.empty())
	{
		short2 island = q.top().island;
		short2 seed = q.top().seed;	
		q.pop();
		if(mapOfIsland[island.y*texSize+island.x].x==texSize)
		{
			mapOfIsland[island.y*texSize+island.x] = seed;

            for (int x = island.x-1; x <= island.x+1; x++)
				for (int y = island.y-1; y <= island.y+1; y++)
					if (x >= min && x < max && y >= min && y < max)
					{
						if (mapOfIsland[texSize * y + x].x == texSize)
						{
							q.push(Node(make_short2(x, y), seed)); 							
						}
					}
		}
	}
}

// Initialize two Pingpong array
void cudaInitializePingpong()
{
    // Compute log2 of texSize
    int tmp = texSize;
    log2Width = 0; 
    while (tmp > 1) { log2Width += 1; tmp >>= 1; }

    // Initialize two color textures with MIN_SHORT
    dim3 block = dim3(BLOCKX, BLOCKY); 
    dim3 grid = dim3(texSize / block.x, texSize / block.y); 
	

    kernelFillShort<<< grid, block >>>(pingpongColor[0], MARKER, log2Width); 
    cutilCheckError(); 
}

// Deallocate all CUDA allocated arrays
void cudaVoronoiDeallocation()
{
    voronoiPtr = (int *) pingpongColor[0]; 
    patternPtr = pingpongPattern[0];

	cutilSafeCall( hipFree(pingpongColor[1]) ); 
//	cutilSafeCall( hipFree(pingpongPattern[1]) ); 
}

// Map points into the texture
// Include scaling and shifting to integer coordinates
void cudaMapPointsToTexture()
{
    dim3 block(WBLOCK); 
    dim3 grid(STRIPE, ROUND(gpudtParams->nPoints, block.x * STRIPE));

    kernelMapPointsToTexture<<< grid, block >>>(gpudtParams->nPoints, cvertices,
        pingpongColor[0], (int *) pingpongPattern[0], log2Width); 
    cutilCheckError(); 
}

// Detect islands
void cudaIslandDetection()
{
    dim3 block = dim3(BLOCKX, BLOCKY); 
    dim3 grid = dim3(texSize / block.x, texSize / block.y); 

    int flag;	

	do 
    {
        // Set flag = 0
        cutilSafeCall( hipMemset(cflag, 0, sizeof(int)) ); 

        cutilSafeCall( hipBindTexture(0, texColor, pingpongColor[iCurrentBuffer], sizeTexture) );  

        kernelIslandDetection<<< grid, block >>>(pingpongColor[1 - iCurrentBuffer], 
            texSize, log2Width, cflag); 

        cutilSafeCall( hipUnbindTexture(texColor) ); 

        // Update the target texture
        iCurrentBuffer = 1 - iCurrentBuffer;
	
		cutilSafeCall( hipMemcpy(&flag, cflag, sizeof(int), hipMemcpyDeviceToHost) ); 

	} while(flag != 0);	

   
    iCurrentBuffer = 1 - iCurrentBuffer;

	// Mark the islands and its neighbors in the list	
	int *islandMark = (int *) pingpongColor[1 - iCurrentBuffer]; 
	int *prefix = (int *) pingpongPattern[1]; 

	cutilSafeCall( hipMemset(islandMark, 0, texSize * texSize * sizeof(int)));	 

	kernelMarkIsland<<< grid, block >>>(pingpongColor[iCurrentBuffer], islandMark, texSize,log2Width);
	cutilCheckError(); 
	
    // Compute the offset of them in the new list
    thrust::exclusive_scan(
        IntDPtr(islandMark), IntDPtr(islandMark) + texSize * texSize,
        IntDPtr(prefix) ); 

	int islands, lastitem; 
    cutilSafeCall( hipMemcpy(&islands, prefix + texSize * texSize - 1, sizeof(int), hipMemcpyDeviceToHost) ); 
    cutilSafeCall( hipMemcpy(&lastitem, islandMark + texSize * texSize - 1, sizeof(int), hipMemcpyDeviceToHost) ); 
    islands += lastitem; 

	short2 *output; 	
	cutilSafeCall( hipMalloc(&output, islands * 2 * sizeof(short2)) );
      
	kernelCollectIsland<<< grid, block >>>(pingpongColor[iCurrentBuffer], islandMark, 
		prefix, output, texSize, log2Width,cflag);
	cutilCheckError();

	short2 *output_cpu = new short2[2*islands];
	cutilSafeCall( hipMemcpy(output_cpu, output, 2*islands * sizeof(short2), hipMemcpyDeviceToHost) ); 
	
	short2 *madeMap = new short2[texSize*texSize];	

	short2 coords, color;
	for(int i=0; i<islands; i++)
	{ 
		coords = output_cpu[i*2    ];		
		color  = output_cpu[i*2 + 1];		
		madeMap[coords.y * texSize + coords.x] = color;		 
	}
   
	colorIsland(madeMap, output_cpu, islands, texSize);	
	
	for(int i=0; i<islands; i++)
	{
		coords = output_cpu[i*2];	
		output_cpu[i*2+1] = madeMap[coords.y * texSize + coords.x];		
	}  

	cutilSafeCall( hipMemcpy(output, output_cpu, 2*islands * sizeof(short2), hipMemcpyHostToDevice) ); 
   
	block = dim3(128); 
    grid = dim3(STRIPE, islands / (STRIPE * block.x) + 1);
	
	kernelRecolorIsland<<< grid, block >>>(pingpongColor[iCurrentBuffer], output, log2Width, islands);
	cutilCheckError(); 	
	
	cutilSafeCall( hipFree(output)); 	
	delete [] output_cpu;
	delete [] madeMap;	
}

// Find real Voronoi vertices
void cudaFindRealVoronoiVertices()
{
    dim3 block = dim3(BLOCKX, BLOCKY); 
    dim3 grid = dim3(texSize / block.x, texSize / block.y); 

    cutilSafeCall( hipBindTexture(0, texInt, pingpongPattern[0], sizeTexture) ); 
    cutilSafeCall( hipBindTexture(0, texColor, pingpongColor[iCurrentBuffer], sizeTexture) ); 

    kernelFindRealVoronoiVertices<<< grid, block >>>(
        (int *) pingpongColor[1 - iCurrentBuffer], pingpongPattern[1],
        texSize, log2Width); 
    cutilCheckError(); 

    cutilSafeCall( hipUnbindTexture(texInt) ); 
    cutilSafeCall( hipUnbindTexture(texColor) ); 

    // Update the target texture
    iCurrentBuffer = 1 - iCurrentBuffer;

    // Make sure we have the latest buffer at buffer 0
    if (iCurrentBuffer != 0)
        cutilSafeCall( hipMemcpy(pingpongColor[0], pingpongColor[1], sizeTexture, 
                   hipMemcpyDeviceToDevice) ); 
}

// Perform prefix sum to link up voronoi vertices in the same row
void cudaPrefixSum()
{
    dim3 block = dim3(BLOCKX, BLOCKY); 
    dim3 grid = dim3(texSize / block.x, texSize / block.y); 

    iCurrentBuffer = 1;        // Working with pattern texture only

    for (int iStepLength = 1; iStepLength < texSize; iStepLength *= 2)
    {
        cutilSafeCall( hipBindTexture(0, texPattern, pingpongPattern[iCurrentBuffer], sizeTexture) ); 

        kernelFlood1D<<< grid, block >>>(pingpongPattern[1 - iCurrentBuffer],
            texSize, log2Width, iStepLength); 

        cutilSafeCall( hipUnbindTexture(texPattern) ); 

        // Update the target texture
        iCurrentBuffer = 1 - iCurrentBuffer;
    }

	cutilCheckError(); 

	// Make sure we have the latest buffer at buffer 0
    if (iCurrentBuffer != 0)
        cutilSafeCall( hipMemcpy(pingpongPattern[0], pingpongPattern[1], sizeTexture, 
                   hipMemcpyDeviceToDevice) ); 
}

/**********************************************************************
 * Construct the discrete voronoi diagram
 **********************************************************************/
void cudaDiscreteVoronoiDiagram() 
{
    // Initialization
    texSize = gpudtParams->fboSize; 

    cudaInitializePingpong(); 
    
    // Map points into the texture
    cudaMapPointsToTexture(); 

	pba2DCompute(phase1Band, phase2Band, phase3Band);
	iCurrentBuffer = 1; 

	cudaIslandDetection(); 
    cudaFindRealVoronoiVertices(); 
    cudaPrefixSum(); 

    // Deallocations
    cudaVoronoiDeallocation(); 

    return ; 
}
