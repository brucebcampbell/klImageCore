#include "hip/hip_runtime.h"
/*
Author: Cao Thanh Tung, Qi Meng
Date: 15/10/2011

File Name: cudaConstraint.cu

This file include all CUDA code to perform the inserting constraints step

===============================================================================

Copyright (c) 2011, School of Computing, National University of Singapore. 
All rights reserved.

Project homepage: http://www.comp.nus.edu.sg/~tants/cdt.html

If you use GPU-DT and you like it or have comments on its usefulness etc., we 
would love to hear from you at <tants@comp.nus.edu.sg>. You may share with us
your experience and any possibilities that we may improve the work/code.

===============================================================================

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution. 

Neither the name of the National University of University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission from the National University of Singapore. 

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES 
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.

*/

#pragma warning(disable: 4311 4312)

#include <hip/device_functions.h>
#include <stdio.h>
#include <string.h>
#include "gpudt.h"
#include "cudaDecl.h"
#include "common.h"
#include "cudaCCW.h"
#include <vector>

using namespace std;

#define MAXINT              2147483647

#define MAX_INNER_LOOP      5

/***********************************************************
* Declarations
***********************************************************/
#define WBLOCK                256  


#define SET_TRIANGLE(vOrg, vDest, vApex, nOrg, nDest, nApex, tri, ori) \
    ctriangles[(tri) * 9 + 3 + ((ori) + 1) % 3] = (vOrg); \
    ctriangles[(tri) * 9 + 3 + ((ori) + 2) % 3] = (vDest); \
    ctriangles[(tri) * 9 + 3 + (ori)] = (vApex); \
    ctriangles[(tri) * 9 + 6 + (ori)] = (nOrg); \
    ctriangles[(tri) * 9 + 6 + ((ori) + 1) % 3] = (nDest); \
    ctriangles[(tri) * 9 + 6 + ((ori) + 2) % 3] = (nApex) 

#define UPDATE_TEMP_LINK(pTriOri, pNext) \
    if ((pTriOri) >= 0) \
    ctriangles[decode_tri(pTriOri) * 9 + 6 + decode_ori(pTriOri)] = -(pNext)

#define UPDATE_LINK(pTriOri, pNext) \
    if ((pTriOri) >= 0) \
    ctriangles[decode_tri(pTriOri) * 9 + decode_ori(pTriOri)] = (pNext)

/**************************************************************
* Exported methods
**************************************************************/
extern "C" void cudaConstraint();

/**************************************************************
* Definitions
**************************************************************/
// Decode an oriented triangle. 
// An oriented triangle consists of 32 bits. 
// - 30 highest bits represent the triangle index, 
// - 2 lowest bits represent the orientation (the starting vertex, 0, 1 or 2)
#define decode_tri(x)            ((x) >> 2)
#define decode_ori(x)            ((x) & 3)
#define encode_tri(tri, ori)    (((tri) << 2) | (ori))

// Encode constraint information for each triangle
// each triangle use 32 bits to record informations as below:
// - 28 highest bits represent the constraint index (-1, if it is not intersected by any constraints; record the maximum index if it is intersected by more than one constriants).
// - 2  represent the orientation (the starting vertex, 0, 1 or 2).
// - 1  represent the apex vertex is on the left/right side of constraint x.
// - 1  represent whether the triangle is the first/last triangle intersected by constraint x.
#define decode_c(x)     (  (x)>>4      )
#define decode_cori(x)  ( ((x)>>2) & 3 )
#define decode_cside(x) ( ((x)>>1) & 1 )
#define decode_clast(x) (  (x) & 1     )

#define encode_constraint(x,ori,side,last)  ( ( ( ( ( (x<<2) | ori) << 1) | side) << 1 ) | last );

// Decode neighbor information for triangle. 
// information of triangle consists of 32 bits. 
// - 30 highest bits represent the neighbor triangle, 
// - 2 lowest bits represent the relationship between triangle and its neighbor
#define decode_neighbor(x)                  ((x) >> 2)
#define decode_candd(x)                     ((x) & 3)
#define encode_neighbor(neighbor, candd)    (((neighbor) << 2) | (candd))

#define MAX(x, y) ((x) < (y) ? (y) : (x))

/************************************************************
* Variables and functions shared with the main module
************************************************************/
extern int nTris, nVerts, nPoints,nConstraints;
extern int *ctriangles;            
extern int *cvertarr;            
extern int *tvertices; 
extern int *cconstraints; 
extern REAL2 *covertices;        
extern short *cnewtri; 
extern int step; 
extern int *cflag; 
extern PGPUDTPARAMS gpudtParams;

/***************************************************************************
* 	For all intersected triangles, find its neighbors which are intersected by the same constraint. 
*	And compute the relationships between triangle and its neighbors; record relationships in CandD. 
***************************************************************************/
__global__ void KernelFlippingPhase_1(int *ctriangles, REAL2 *covertices, int *cconnection, 
                                      int nTris, int *cflag, int *flipBy, BYTE* doNot, BYTE *affect, 
									  int timestamp, int *active, int noActive)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (x >= noActive) 
        return; 

	if ( x == 0 ) 
		*cflag = 0; 

	x = active[x]; 

	flipBy[x] = -1; 
    doNot[x] = -1;

    if (affect[x] != 1) 
        return;

	int cc = cconnection[x]; 

    if (cc < 0)
        return;

	int intersectby = decode_c(cc);
    int ori			= decode_cori(cc); 
    int side		= decode_cside(cc); 
    int last		= decode_clast(cc); 	

    int nRight;	

    if(side==1)
        nRight = ctriangles[x*9 + (ori+2)%3];
    else
        nRight = ctriangles[x*9 + (ori+1)%3];

    if(nRight<0)
        return;

    REAL2 p1, p2, p3, p4;	
    int pTri, pOri;
    REAL test1, test2;	
    int side_pTri, last_pTri;		

    pTri = decode_tri(nRight);	pOri = decode_ori(nRight);
	int cc_pTri = cconnection[pTri]; 
    int intersectby_pTri = decode_c(cc_pTri) ;

	if ( intersectby != intersectby_pTri ) 
		return ; 

    int configuration = 0;//initial as single/zero configuration

    //check whether there's concave situation
    p1 = covertices[ctriangles[x*9 + 3 + (ori+2)%3]]; 
    p2 = covertices[ctriangles[x*9 + 3 + (ori+1)%3]]; 
    p3 = covertices[ctriangles[x*9 + 3 +  ori     ]]; 
    p4 = covertices[ctriangles[pTri * 9 + 3 + pOri]];		

	if ( side == 0 ) 
	{
		REAL2 tmp;
		tmp = p1; p1 = p2; p2 = tmp; 
	}

    test1 = cuda_ccw(p1, p4, p2);
    test2 = cuda_ccw(p1, p4, p3);

    if (test1*test2 >= 0)   // concave		
        configuration  = 1;
    else
    {
        //check whether there's double-intersection situation
        side_pTri = decode_cside(cc_pTri);
        last_pTri = decode_clast(cc_pTri);
        if(side_pTri != side && last!=1 && last_pTri!=1)//double-intersection
            configuration  = 2;			
    }		

	doNot[x] = configuration; 
}

/***************************************************************************
* 	For all intersected triangles, check whether it satisfies the 4 cases with its neighbors. 
*	Record this information in doNot. 
***************************************************************************/

__global__ void KernelFlippingPhase_2(int *ctriangles, REAL2 *covertices, int *cconnection, int nTris,
                                     BYTE *doNot, BYTE *affect, int *flipBy, int *cflag, BYTE *flipOri, 
									 int timestamp, int *active, int noActive)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= noActive ) 
        return ; 

	x = active[x]; 

    if (affect[x] == 0)
		return ; 

	affect[x] = 0; 

    int cc = cconnection[x];
    if( cc < 0)
        return;	

    int side = decode_cside(cc);
    int ori = decode_cori(cc);
    int previous = ctriangles[x*9 + (ori+1)%3]; 
    int next = ctriangles[x*9 + (ori+2)%3]; 

    if(side==0)
    {
        int temp = previous;
        previous = next;
        next = temp;
    }

    if(next<0)
        return;

    int pRight = decode_tri(next);
    int constraintsforx = decode_c(cc);
    int candd_right = doNot[x];

	int isFlippable = 0; 

    if(candd_right==0)	//case1
    { 
        atomicMax(&flipBy[x], x+(1<<30));              
        atomicMax(&flipBy[pRight], x+(1<<30));							
        flipOri[x] = (ori+side)%3;			
	    *cflag = 1;	

		return ; 
    } 

    if(previous<0)
        return;

    int pLeft = decode_tri(previous);
    int constraintsforleft = decode_c(cconnection[pLeft]);

	if ( constraintsforleft != constraintsforx ) 
		return ; 

    int candd_left = doNot[pLeft];

    if(    (candd_left==2 &&  candd_right==2)	//case2
		|| (candd_left==1 &&  candd_right==2) )	//case3
    { 
        int pLeftOri = decode_ori(previous);
        int pRightOri = decode_ori(next);

        REAL2 pApex = covertices[ctriangles[pLeft    * 9 + 3 + pLeftOri]]; 
        REAL2 pOrg  = covertices[ctriangles[pLeft    * 9 + 3 + (pLeftOri+1)%3]]; 
        REAL2 pDest = covertices[ctriangles[pLeft    * 9 + 3 + (pLeftOri+2)%3]]; 
        REAL2 pOpp  = covertices[ctriangles[pRight   * 9 + 3 + pRightOri]]; 

        REAL test1 = cuda_ccw(pOpp,pApex,pOrg);
        REAL test2 = cuda_ccw(pOpp,pApex,pDest);		

        if(test1*test2<0)	
			isFlippable = ( candd_left == 2 ? 2 : 3 );
		else
			return ; 

        int shift;

        if(isFlippable==2)
            shift = 29;
        else
            shift = 28;		

        atomicMax(&flipBy[x],       x + (1 << shift));
        atomicMax(&flipBy[pLeft],   x + (1 << shift));
        atomicMax(&flipBy[pRight],  x + (1 << shift));

        flipOri[x] = (ori+side)%3;	
	    *cflag = 1;	
    }
}

/***************************************************************************
* 	For each triangle, record the intersection information with constraints.
***************************************************************************/
__global__ void kernelCheckingPhase(int *ctriangles, int *cvertarr, int *cconnection, REAL2 *covertices, 
                                    int nConstraints, int *tconstraintLink_flip, BYTE* done)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x+threadIdx.x;

    if (x >= nConstraints || done[x] == 1) 
        return ;

    int t = 0;    
    int point1 = tconstraintLink_flip[2*x];
    int point2 = tconstraintLink_flip[2*x+1];
    int pTri, pOri, pTri9, pApex, pDest;
    int neighbor;
    int p = cvertarr[point1];

    int nOpp, pnext;
    pTri = decode_tri(p);
    pOri = decode_ori(p);
    pApex = ctriangles[pTri*9 + 3 +  pOri];
    pDest = ctriangles[pTri*9 + 3 +  (pOri+2)%3];

    if(pApex == point2 || pDest==point2)
	{
		// Mark the constraint so we don't check it again next rounds
		done[x] = 1; 
        return; 
	}

    REAL2 point1_R,point2_R,point3_R,point4_R;
    point1_R = covertices[point1];
    point2_R = covertices[point2];

    point3_R = covertices[pApex];
    REAL test = cuda_ccw(point1_R,point2_R,point3_R);

    if(test<0)//pApex is on the right of direction point1point2, so we need to find leftside pTri*9+2
    {	
        pnext = ctriangles[pTri*9 + (2 + pOri)%3];
        //find the first triangle that intersection constraint x.		
        do
        {			
            pTri = decode_tri(pnext);
            pOri = decode_ori(pnext);		
            pApex = ctriangles[pTri*9 + 3 + (0+pOri)%3];	
            if(pApex==point2) //pOrg=point1, pApex=point2;
			{
				done[x] = 1; 
                return;
			}
            point4_R = covertices[pApex];
            test = cuda_ccw(point1_R,point2_R,point4_R);

            if( test > 0)
            {
                nOpp = ctriangles[pTri*9 + (1 + pOri)%3];
                t = encode_constraint(x,pOri,0,1);  //ori=pOri,side=0,last=1;	
                atomicMax(&cconnection[pTri],t);
                break; 
            }
            p = pnext;		
            pnext = ctriangles[pTri*9 + (2 + pOri)%3];

        } while (true);
    }
    else//pApex is on the left of direction point1point2, so we need to find rightside pTri*9+0
    {		
        pnext = cvertarr[point1];		
        //find the first triangle that intersection constraint x.	
        do
        {
            if(pDest==point2) //pOrg=point1, pDest=point2;
			{
				done[x] = 1; 
 				return;
			}
            point4_R = covertices[pDest];
            test = cuda_ccw(point1_R,point2_R,point4_R);			

            if( test < 0 )
            {			
                nOpp = ctriangles[pTri*9 + (1 + pOri)%3];
                t = encode_constraint(x,pOri,0,1);  //ori=pOri,side=0,last=1;	
                atomicMax(&cconnection[pTri],t);	          
                break; 
            }
            p = pnext;	
            pnext = ctriangles[pTri*9 + (0+pOri)%3];	
            pTri = decode_tri(pnext);
            pOri = ( decode_ori(pnext) + 1 ) % 3;	
			pDest = ctriangles[pTri*9 + 3 + (2+pOri)%3]; 	         
        } while (true);
    }

    //find other triangles that intersected by constraint x.	
    pnext = nOpp;	
    int ori, side, last;
    side = 0;
    int tp;
    do{		
        pTri = decode_tri(pnext);
        pOri = decode_ori(pnext);
        pTri9 = pTri*9;	
        tp = ctriangles[pTri9 + 3 + pOri]; 

        if(tp==point2)
        {
            side = 1;
            ori = (pOri+2)%3;			
            t = encode_constraint(x, ori, side, 1);
            atomicMax(&cconnection[pTri], t);			
            return;
        }
        point3_R = covertices[tp];
        test = cuda_ccw(point1_R, point2_R, point3_R);

        if(test<0)//pApex (tp) is on the rightside of the constraint.
        {   
            ori = (pOri + 1)%3;
            side = 0;
            last = 0;
            neighbor = ctriangles[pTri9 + (pOri+2)%3];		
            t = encode_constraint(x, ori, side, last);
            atomicMax(&cconnection[pTri], t);			
            pnext = neighbor;			
        }
        if(test >0)///pApex (tp) is on the leftside of the constraint.
        {			
            ori = (pOri + 2)%3;
            side = 1;			
            last = 0;
            neighbor = ctriangles[pTri9 + (pOri+1)%3];
            t = encode_constraint(x, ori, side, last);
            atomicMax(&cconnection[pTri], t);				
            pnext = neighbor;					
        }
    }while(true);
}

// To flip a pair of triangle, we need to update the links between neighbor
// triangles. There is a chance that we need to flip two pairs in which 
// there are two neighboring triangles, thus the flipping is performed
// in 3 phases: 
// - Phase 1: Update the triangle vertices. Also, note down the new 
//            neighbors of each triangles in the 3 nexttri links (previously
//            used to store the vertex array). 
// - Phase 2: Each new triangle will then inform its new neighbors of its 
//            existence by writing itself in the corresponding nexttri link
//            of its neighbor.
// - Phase 3: Each new triangle update its links to its neighbors using the
//            3 new nexttri links. If any of the link is not updated, that
//            means the corresponding neighbor is not flipped in this pass. 
//            We then actively update the corresponding link in that triangle.

__global__ void kernelUpdatePhase11(int *ctriangles, BYTE *cflipOri, int *cflipStep, 
                                    int *cflipBy, int nTris, int step, int *cvertarr, 
                                    int *cconnection, BYTE *affect, BYTE *doNot, int *active, int noActive)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= noActive)
        return ;

	x = active[x]; 

    int cc = cconnection[x];
    if(cc<0)
        return;		

    int xori, xside, xid, xlast, pori, pside, plast;
    int ori, pOpp, pOppTri, pOppOri, pOrg, pDest, pApex, nOrg, nApex, npDest, npApex;
    int newtag = -1; 
    int pLeft, pRight, pLeftTri, pRightTri;

    if ((cflipBy[x]&0xFFFFFFF) == x) 
    {     
        // I'm the one who win the right to flip myself

        xid   = decode_c(cc);
        xori  = decode_cori(cc);
        xside = decode_cside(cc);
        xlast = decode_clast(cc); 

        if(xside==0)	
            pLeft = ctriangles[x * 9 + (xori+2)%3];
        else	
            pLeft = ctriangles[x * 9 + (xori+1)%3];	

        ori     = cflipOri[x]; 
        pOpp    = ctriangles[x * 9 + (ori + 1) % 3]; 	
        pOppTri = decode_tri(pOpp);
        pOppOri = decode_ori(pOpp); 
        int pcc = cconnection[pOppTri];
        bool flip = false;

        if(cflipBy[x]>=(1 << 30) )
        {
            if((cflipBy[pOppTri]&0xFFFFFFF) == x)
                flip = true;
        }
        else
        {
            if(pLeft>-1)
            {
                pLeftTri = decode_tri(pLeft);
                if((cflipBy[pLeftTri]&0xFFFFFFF) == x && (cflipBy[pOppTri]&0xFFFFFFF) == x)
                    flip = true;
            }
        }
        if(flip)
        {    // I'm also the one who win the right

            newtag = pOpp;              // to flip this neighbor, so I can flip.
            pOrg = ctriangles[x * 9 + 3 + (ori + 1) % 3]; 
            pDest = ctriangles[x * 9 + 3 + (ori + 2) % 3]; 
            pApex = ctriangles[x * 9 + 3 + ori];
            pOpp = ctriangles[pOppTri * 9 + 3 + pOppOri]; 
            nOrg = ctriangles[x * 9 + ori]; 
            nApex = ctriangles[x * 9 + (ori + 2) % 3]; 
            npDest = ctriangles[pOppTri * 9 + (pOppOri + 1) % 3]; 
            npApex = ctriangles[pOppTri * 9 + (pOppOri + 2) % 3]; 

            // Update vertices + nexttri links
            SET_TRIANGLE(pOrg, pDest, pOpp, 3 + nOrg, -1, 3 + nApex, x, ori); 
            SET_TRIANGLE(pApex, pOrg, pOpp, -1, 3 + npDest, 3 + npApex, pOppTri, pOppOri); 

            //update cvertarr
            cvertarr[pOrg] = (x<<2)|((ori+1+2)%3);
            cvertarr[pDest] = (x<<2)|((ori+2+2)%3);
            cvertarr[pApex] = (pOppTri<<2)|((pOppOri+1+2)%3);
            cvertarr[pOpp] = (x<<2)|((ori+0+2)%3);

            pside = decode_cside(pcc);
            pori = decode_cori(pcc);
            plast = decode_clast(pcc);

            if(pside==0)
                pRight = ctriangles[pOppTri * 9 + (pori+1)%3];	 	 
            else
                pRight = ctriangles[pOppTri * 9 + (pori+2)%3];	

            affect[x] = 1;
            affect[pOppTri] = 1;	
            if(pRight>-1)
            {
                pRightTri = decode_tri(pRight);
                int c = decode_c(cconnection[pRightTri]);
                if(xid==c)
                {		
                    affect[pRightTri] = 1;			
                }
            }
            if(pLeft>-1)
            {
                pLeftTri = decode_tri(pLeft);
                int c = decode_c(cconnection[pLeftTri]);
                if(xid==c)
                {			
                    affect[pLeftTri] = 1;			
                }
            }

            ///update cconnection array
            pside = decode_cside(pcc);
            pori = decode_cori(pcc);
            plast = decode_clast(pcc);
            int sOrg,sDest,sApex,sOpp;
            if( ((ori + 1) % 3) == xori ) sOrg = xside; else sOrg = 1-xside;
            if( ((ori + 2) % 3) == xori ) sDest = xside;else sDest = 1-xside;
            if( ((ori    ) % 3) == xori ) sApex = xside;else sApex = 1-xside;
            if( pOppOri == pori) sOpp = pside;else sOpp = 1-pside;		

            if(xlast==1 && plast==1)
            {
                cconnection[x] = -1;
                cconnection[pOppTri] = -1;	

            }
            else if(xlast==0 && plast==0)
            {
                if (sOrg==sOpp)//only one triangle is intersected after flip
                {
                    if(sOrg == sDest)// x is not intersected
                    {
                        cconnection[x] = -1;
                        pori = (pOppOri+1)%3;
                        pside = sApex;
                        cconnection[pOppTri] = encode_constraint(xid,pori,pside,plast);
                    }
                    else// x is intersected
                    {
                        cconnection[pOppTri] = -1;						
                        xori = (ori+2)%3;
                        xside = sDest;
                        cconnection[x] = encode_constraint(xid,xori,xside,xlast);

                    }
                }
                else// both triangles are intersected after flip
                { 
                    if(sOrg ==sDest)
                    {
                        xori = ori;
                        xside = sOpp;						
                        pori = (pOppOri+2)%3;
                        pside = sOrg;						
                    }
                    else
                    {						
                        xori = (ori+1)%3;
                        xside = sOrg;					
                        pori = (pOppOri+0)%3;
                        pside = sOpp;
                    }
                    cconnection[x] = encode_constraint(xid,xori,xside,xlast);
                    cconnection[pOppTri] = encode_constraint(xid,pori,pside,plast);
                }
            }
            else if(xlast==1 && plast==0)
            {
                if(sDest == sOpp)// x is not intersected
                {
                    cconnection[x] = -1;					
                    pori = (pOppOri+1)%3;
                    pside = sApex;
                    plast = 1;
                    cconnection[pOppTri] = encode_constraint(xid,pori,pside,plast);
                }
                else// x is intersected
                {
                    cconnection[pOppTri] = -1;				
                    xori = (ori+2)%3;
                    xside = sDest;
                    xlast = 1;
                    cconnection[x] = encode_constraint(xid,xori,xside,xlast);
                }
            }
            else//xlast==0 && plast==1
            {
                if(sOrg == sDest)//x is not intersected
                {
                    cconnection[x] = -1;				
                    pori = (pOppOri+1)%3;
                    pside = sApex;
                    plast = 1;
                    cconnection[pOppTri] = encode_constraint(xid,pori,pside,plast);
                }
                else// x is intersected
                {
                    cconnection[pOppTri] = -1;			
                    xori = (1+ori)%3;
                    xside = sOrg;
                    xlast = 1;
                    cconnection[x] = encode_constraint(xid,xori,xside,xlast);
                }
            }
        }
    }	

    // Record the opp triangle to be used in the next phase
    cflipStep[x] = newtag;  
}

__global__ void kernelUpdatePhase22(int *ctriangles, BYTE *cflipOri, int *cflipStep, int nTris, int *active, int noActive) 
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if (x >= noActive)
		return; 

	x = active[x]; 

    if (cflipStep[x] < 0) 
        return ; 

    int ori, pOpp, pOppTri, pOppOri, nOrg, nApex, npDest, npApex;

    ori = cflipOri[x]; 
    pOpp = cflipStep[x]; 
    pOppTri = decode_tri(pOpp);
    pOppOri = decode_ori(pOpp); 
    nOrg = ctriangles[x * 9 + ori]; 
    nApex = ctriangles[x * 9 + (ori + 2) % 3]; 
    npDest = ctriangles[pOppTri * 9 + (pOppOri + 1) % 3]; 
    npApex = ctriangles[pOppTri * 9 + (pOppOri + 2) % 3]; 

    // Update my neighbors of my existence
    UPDATE_TEMP_LINK(nOrg, encode_tri(x, ori)); 
    UPDATE_TEMP_LINK(nApex, encode_tri(pOppTri, pOppOri)); 
    UPDATE_TEMP_LINK(npDest, encode_tri(x, (ori + 1) % 3)); 
    UPDATE_TEMP_LINK(npApex, encode_tri(pOppTri, (pOppOri + 2) % 3)); 
}

__global__ void kernelUpdatePhase33(int *ctriangles, BYTE *cflipOri, int *cflipStep, 
                                    int nTris, int *active, int noActive) 
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if (x >= noActive)
		return ;

	x = active[x]; 

    if (cflipStep[x] < 0) 
        return ; 

    int ori, pOpp, pOppTri, pOppOri, nOrg, nApex, npDest, npApex;

    ori = cflipOri[x]; 
    pOpp = cflipStep[x]; 
    pOppTri = decode_tri(pOpp);
    pOppOri = decode_ori(pOpp); 

    // Update other links
    nOrg = ctriangles[x * 9 + 6 + ori]; 
    nApex = ctriangles[x * 9 + 6 + (ori + 2) % 3]; 
    npDest = ctriangles[pOppTri * 9 + 6 + (pOppOri + 1) % 3]; 
    npApex = ctriangles[pOppTri * 9 + 6 + (pOppOri + 2) % 3]; 

    if (nOrg > 0) {        // My neighbor do not update me, update him
        nOrg = -(nOrg - 3); 
        UPDATE_LINK(-nOrg, encode_tri(x, ori)); 
    }

    if (nApex > 0) {
        nApex = -(nApex - 3); 
        UPDATE_LINK(-nApex, encode_tri(pOppTri, pOppOri)); 
    }

    if (npDest > 0) {
        npDest = -(npDest - 3); 
        UPDATE_LINK(-npDest, encode_tri(x, (ori + 1) % 3)); 
    }

    if (npApex > 0) {
        npApex = -(npApex - 3); 
        UPDATE_LINK(-npApex, encode_tri(pOppTri, (pOppOri + 2) % 3)); 
    }

    // Update my own links
    ctriangles[x * 9 + ori] = -nOrg; 
    ctriangles[x * 9 + (ori + 1) % 3] = -npDest; 
    ctriangles[x * 9 + (ori + 2) % 3] = encode_tri(pOppTri, (pOppOri + 1) % 3); 
    ctriangles[pOppTri * 9 + pOppOri] = -nApex; 
    ctriangles[pOppTri * 9 + (pOppOri + 1) % 3] = encode_tri(x, (ori + 2) % 3); 
    ctriangles[pOppTri * 9 + (pOppOri + 2) % 3] = -npApex; 

    // Mark the affected triangles, so that we will check again in the next pass
    cflipStep[x] = -1;   
}

/********************************************************************
* Fix vertex array
********************************************************************/

//__global__ void kernelFixVertArray(int *ctriangles, int nTris, int *cvertarr) 
//
//{
//    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;	
//    if (x >= nTris)
//        return ; 
//
//    int v0 = ctriangles[x * 9 + 4];
//    int v1 = ctriangles[x * 9 + 5];
//    int v2 = ctriangles[x * 9 + 3];
//
//    ctriangles[x * 9 + 6] = atomicExch(&cvertarr[v0], (x << 2)); 
//    ctriangles[x * 9 + 7] = atomicExch(&cvertarr[v1], (x << 2) | 1); 
//    ctriangles[x * 9 + 8] = atomicExch(&cvertarr[v2], (x << 2) | 2); 
//}
//
__global__ void kernelSetIsActive(int *active, int *cconnection, int noActive)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x+threadIdx.x;

    if (x >= noActive) 
        return ;

	if ( cconnection[ active[x] ] < 0 )
		active[x] = -1; 
}

__global__ void kernelInitialize(int *active, BYTE *affect, int *cconnection, int noActive)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x+threadIdx.x;

    if (x >= noActive) 
        return ;

	x = active[x]; 

	affect[x] = 1; 
	cconnection[x] = -1; 
}

/********************************************************************
* Insert constraints 
********************************************************************/
void cudaConstraint()
{
    if (nConstraints == 0)
        return;

    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(constData), hostConst, 13 * sizeof(REAL)) ); 	

    dim3 block = dim3(128);
    dim3 grid = dim3(STRIPE, nTris/(STRIPE * block.x) + 1);

    int *cconnection, *active; 
    int flag_inner = 0, timestamp = 0;		
    int *cflipStep, *cflipBy;
    BYTE  *cflipOri;	
    BYTE *doNot;
    BYTE *affect;
	BYTE *done; 

	cutilSafeCall( hipMalloc( (void**)&done, nConstraints ) ); 
    cutilSafeCall( hipMalloc( (void**)&cconstraints,   2 * nConstraints    * sizeof(int) ) );
    cutilSafeCall( hipMalloc( (void**)&cconnection,    nTris               * sizeof(int) ));
    cutilSafeCall( hipMalloc( (void**)&active,    		nTris               * sizeof(int) ));
    cutilSafeCall( hipMalloc( (void**)&doNot,          nTris               * sizeof(BYTE) )); 
    cutilSafeCall( hipMalloc( (void**)&cflipStep,      nTris               * sizeof(int) ));
    cutilSafeCall( hipMalloc( (void**)&cflipBy,        nTris               * sizeof(int) ));
    cutilSafeCall( hipMalloc( (void**)&cflipOri,       nTris               * sizeof(BYTE)));		 
    cutilSafeCall( hipMalloc( (void**)&affect,         nTris               * sizeof(BYTE)));

	IntDPtr activePtr( active ); 
	int noActive = nTris;  

	thrust::sequence( activePtr, activePtr + nTris ); 

    cutilSafeCall( hipMemset(cflipStep, 255, nTris * sizeof(int)) );	
	cutilSafeCall( hipMemset(done, 0, nConstraints) ); 

    // Copy constraints from host to device. 
    // This is also used in cudaFlipping, and will be released there. 
    cutilSafeCall( hipMemcpy(cconstraints, gpudtParams->constraints, nConstraints * 2 * sizeof(int), hipMemcpyHostToDevice) );

    int step = 0;

	do 
    { 	
		dim3 bigBlock = dim3(256);
		grid = dim3(STRIPE, noActive/(STRIPE * bigBlock.x) + 1); 
		kernelInitialize<<< grid, bigBlock >>>(active, affect, cconnection, noActive); 
		cutilCheckError(); 

        // outer-loop. Checking for all constraints, find all intersected triangles.

        grid = dim3(STRIPE, nConstraints/(STRIPE * block.x) + 1);		
        kernelCheckingPhase<<< grid, block >>>(ctriangles, cvertarr,  cconnection,covertices, nConstraints, cconstraints, done); 			
        cutilCheckError();

		grid = dim3(STRIPE, noActive/(STRIPE * bigBlock.x) + 1); 
		kernelSetIsActive<<< grid, bigBlock >>>(active, cconnection, noActive); 
		cutilCheckError(); 

		IntDPtr lastActivePtr = thrust::remove_if( activePtr, activePtr + noActive, isNegative() ); 

		noActive = lastActivePtr - activePtr; 

        if (noActive > 0) 
        {
            //inner-loop. Find all flippable triangle pairs and flip them until there's no flippalbe triangle pairs left.
            do
            {
                timestamp++;				
                grid = dim3(STRIPE, noActive/(STRIPE * block.x) + 1);	

                //For all intersected triangles, find its neighbors which are intersected by the same constraint. 
                //And compute the relationships between triangle and its neighbors; record relationships in doNot.
                KernelFlippingPhase_1<<< grid, block >>>(ctriangles, covertices, cconnection, nTris, cflag, cflipBy, 
					doNot, affect,timestamp, active, noActive);
                cutilCheckError();

                // For all intersected triangles, check whether it can be flipped with its neighbors.		
                KernelFlippingPhase_2<<< grid, block >>>(ctriangles, covertices, cconnection, nTris, doNot, affect, 
					cflipBy, cflag, cflipOri, timestamp, active, noActive);
                cutilCheckError();
                cutilSafeCall( hipMemcpy(&flag_inner, cflag, sizeof(int), hipMemcpyDeviceToHost) ); 

                if(flag_inner>0)				
                {
                    // do flipping on flippable triangle pairs, and update link information between triangles.
                    kernelUpdatePhase11<<< grid, block >>>(ctriangles, cflipOri, cflipStep, cflipBy, nTris, 
                        timestamp, cvertarr, cconnection, affect, doNot, active, noActive);				

                    kernelUpdatePhase22<<< grid, block >>>(ctriangles, cflipOri, cflipStep,	nTris, active, noActive);  
                    kernelUpdatePhase33<<< grid, block >>>(ctriangles, cflipOri, cflipStep,	nTris, active, noActive); 	
                }	

                // We only run a few inner loops for each outer loop
                // After that, the number of flippable pairs are too small.
                if (timestamp % MAX_INNER_LOOP == 0)
                    flag_inner = 0;		
            } while (flag_inner > 0);
        }	
        step ++;	

    } while ( noActive > 0 );

	cutilSafeCall( hipFree(done) );
	cutilSafeCall( hipFree(active) ); 
    cutilSafeCall( hipFree(cconnection)); 	
    cutilSafeCall( hipFree(cflipOri));
    cutilSafeCall( hipFree(cflipBy));
    cutilSafeCall( hipFree(cflipStep));
    cutilSafeCall( hipFree(doNot));	
    cutilSafeCall( hipFree(affect));  
}
