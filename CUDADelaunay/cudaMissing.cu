#include "hip/hip_runtime.h"
/*
Author: Cao Thanh Tung, Qi Meng
Date: 15/03/2011

File Name: cudaMissing.cu

This file include all CUDA code to perform the inserting missing sites step

===============================================================================

Copyright (c) 2011, School of Computing, National University of Singapore. 
All rights reserved.

Project homepage: http://www.comp.nus.edu.sg/~tants/cdt.html

If you use GPU-DT and you like it or have comments on its usefulness etc., we 
would love to hear from you at <tants@comp.nus.edu.sg>. You may share with us
your experience and any possibilities that we may improve the work/code.

===============================================================================

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution. 

Neither the name of the National University of University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission from the National University of Singapore. 

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES 
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.

*/

#pragma warning(disable: 4311 4312)

#include <hip/device_functions.h>
#include <stdio.h>
#include <string.h>
#include "gpudt.h"
#include "cudaDecl.h"
#include "common.h"
#include "cudaCCW.h"

#define MAXINT        2147483647

/***********************************************************
 * Declarations
 ***********************************************************/
#define WBLOCK                256  
#define INSERT_TRIANGLE(v0, v1, v2, tri) \
    tmp = tri; \
    cnewtri[tmp] = step; \
    ctriangles[tmp * 9 + 3] = v2; \
    ctriangles[tmp * 9 + 4] = v0; \
    ctriangles[tmp * 9 + 5] = v1; \
    ctriangles[tmp * 9 + 6] = atomicExch(&cvertarr[v0], (tmp << 2)); \
    ctriangles[tmp * 9 + 7] = atomicExch(&cvertarr[v1], (tmp << 2) | 1); \
    ctriangles[tmp * 9 + 8] = atomicExch(&cvertarr[v2], (tmp << 2) | 2); \

#define SET_TRIANGLE(vOrg, vDest, vApex, nOrg, nDest, nApex, tri, ori) \
	ctriangles[(tri) * 9 + 3 + ((ori) + 1) % 3] = (vOrg); \
	ctriangles[(tri) * 9 + 3 + ((ori) + 2) % 3] = (vDest); \
	ctriangles[(tri) * 9 + 3 + (ori)] = (vApex); \
	ctriangles[(tri) * 9 + 6 + (ori)] = (nOrg); \
	ctriangles[(tri) * 9 + 6 + ((ori) + 1) % 3] = (nDest); \
	ctriangles[(tri) * 9 + 6 + ((ori) + 2) % 3] = (nApex) 

#define UPDATE_TEMP_LINK(pTriOri, pNext) \
	if ((pTriOri) >= 0) \
	ctriangles[decode_tri(pTriOri) * 9 + 6 + decode_ori(pTriOri)] = -(pNext)

#define UPDATE_LINK(pTriOri, pNext) \
	if ((pTriOri) >= 0) \
	ctriangles[decode_tri(pTriOri) * 9 + decode_ori(pTriOri)] = (pNext)

/**************************************************************
 * Exported methods
 **************************************************************/
extern "C" void cudaMissing();

/**************************************************************
 * Definitions
 **************************************************************/
// Decode an oriented triangle. 
// An oriented triangle consists of 32 bits. 
// - 30 highest bits represent the triangle index, 
// - 2 lowest bits represent the orientation (the starting vertex, 0, 1 or 2)
#define decode_tri(x)            ((x) >> 2)
#define decode_ori(x)            ((x) & 3)
#define encode_tri(tri, ori)    (((tri) << 2) | (ori))


#define MAX(x, y) ((x) < (y) ? (y) : (x))

/************************************************************
 * Variables and functions shared with the main module
 ************************************************************/
extern int nTris, nVerts, nPoints,nConstraints;       
extern int *ctriangles;            
extern int *cvertarr;            
extern int *tvertices; 
extern REAL2 *covertices;        
extern short *cnewtri; 
extern int step; 
extern int *cflag; 


/*******************************************************************
 * Fill an array with increasing numbers
 *******************************************************************/
__global__ void kernelFillIncrement(int *list, int start, int length) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
	int noThreads = blockDim.x * gridDim.x; 

	for (; x < length; x += noThreads)
	    list[x] = start + x; 
}


/********************************************************************
 * Collect all dead triangles into a list. 
 ********************************************************************/
__global__ void kernelMarkDeadTriangles(int *cmarker, short *cnewtri, int nTris) {
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris)
        return ; 

    cmarker[x] = (cnewtri[x] >= 0 ? 0 : 1); 
}

__global__ void kernelCollectDeadTriangles(int *cdeadTri, short *cnewtri, int *cmarker, int nTris) {
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris || cnewtri[x] >= 0)
        return ; 

    int id = cmarker[x]; 

    cdeadTri[id] = x; 
}

/********************************************************************
 * Locate the triangle which we are gonna insert a missing site to. 
 * - If the anchor of the missing site is not yet inserted, skip
 * - Locate the triangle and mark it to avoid two insertions into 
 *   the same triangle. 
 * - Guarantee that the missing site is not on the boundary due to 
 *   huge fake boundary added. 
 ********************************************************************/
__global__ void kernelLocateTriangleContainer(int *ctriangles, int *cvertarr, int *tvertices, 
                                              int *clocation, int *ctags, REAL2 *covertices, 
                                              int nVerts, int *active, int noActive)
{
    int t = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if ( t >= noActive )
        return ;

    int x = active[t]; 

    int anchor = tvertices[x]; 

    if (cvertarr[anchor] < 0) {
        clocation[x] = MAXINT; 
        return ; 
    }

    REAL2 v = covertices[x]; 
    REAL2 vanchor = covertices[anchor]; 

    REAL ccDest, ccApex, ccOpposite; 
    int pNextTri, pTri, pOri; 

    pNextTri = cvertarr[anchor]; 

    pTri = decode_tri(pNextTri);
    pOri = decode_ori(pNextTri); 

    int pDest, pApex, pOrg, pTmp; 
    REAL2 vDest, vApex, vOrg, vTmp; 

    pDest = ctriangles[pTri * 9 + 3 + (pOri + 2) % 3];    // Dest
    vDest = covertices[pDest]; 

    ccDest = cuda_ccw(vanchor, vDest, v); 

    do {
        pApex = ctriangles[pTri * 9 + 3 + pOri];        // apex
        vApex = covertices[pApex]; 

        ccApex = cuda_ccw(vanchor, vApex, v); 

        if (ccDest >= 0.0 && ccApex <= 0.0)
            // Inside the angle
            break; 

        pDest = pApex; vDest = vApex; ccDest = ccApex; 

        pNextTri = ctriangles[pTri * 9 + (pOri + 2) % 3]; 

        pTri = decode_tri(pNextTri);
        pOri = decode_ori(pNextTri); 
    } while (true);

    // Found an angle, now look for the actual triangle 
    // containing me. 
    ccOpposite = cuda_ccw(vDest, vApex, v);
    if (ccOpposite < 0.0) {
        // It's not right here, need to walk a bit further
        while (true) {
            // Get the opposite triangle
            pNextTri = ctriangles[pTri * 9 + (pOri + 1) % 3]; 
            //if (pNextTri < 0) {
            //    cvertarr[x] = -100; 
            //    clocation[x] = encode_tri(pTri, (pOri + 1) % 3); 
            //    return ; 
            //}
            pTri = decode_tri(pNextTri);
            // Rotate the triangle so that the org is opposite the previous org
            pOri = (decode_ori(pNextTri) + 2) % 3;    
            
            pOrg = ctriangles[pTri * 9 + 3 + (pOri + 1) % 3]; 
            vOrg = covertices[pOrg]; 

            pTmp = pDest; pDest = pApex; pApex = pTmp; 
            vTmp = vDest; vDest = vApex; vApex = vTmp; 

            ccDest = cuda_ccw(vOrg, vDest, v); 
            ccApex = cuda_ccw(vApex, vOrg, v); 

            bool moveleft; 

            if (ccDest >= 0.0)
                if (ccApex >= 0.0) 
                    // Found it!
                    break; 
                else
                    moveleft = false; 
            else
                if (ccApex >= 0.0)
                    moveleft = true; 
                else 
                    moveleft = (vOrg.x - v.x) * (vApex.x - vDest.x) + 
                               (vOrg.y - v.y) * (vApex.y - vDest.y) > 0.0; 

            if (moveleft) {
                pOri = (pOri + 2) % 3; 
                pApex = pDest; pDest = pOrg; 
                vApex = vDest; vDest = vOrg; 
                ccOpposite = ccDest;    // Orientation is unimportant
            }
            else {
                pOri = (pOri + 1) % 3; 
                pDest = pApex; pApex = pOrg; 
                vDest = vApex; vApex = vOrg; 
                ccOpposite = ccApex; 
            }
        }
    }

    int c0 = 0; 

    if (ccDest == 0.0) c0++; 
    if (ccApex == 0.0) c0++; 
    if (ccOpposite == 0.0) c0++; 

    if (c0 == 0) {
        // Easiest case, it's right here!
        clocation[x] = pNextTri + 1;    // Mark to indicate that it's a simple case.
        atomicMin(&ctags[pTri], x); 
    } else if (c0 > 1) {
        // Duplicate point
        clocation[x] = pNextTri + 1;
        cvertarr[x] = -2;
        active[t] = -1; 
        return ; 
    } else {
        // On an edge. 
        // Make sure our 'location' triangle always face toward that edge
        // (i.e. that edge will be opposite to the origin)
        if (ccDest == 0.0) 
            pOri = (pOri + 2) % 3; 
        else if (ccApex == 0.0) 
            pOri = (pOri + 1) % 3; 

        clocation[x] = -encode_tri(pTri, pOri) - 1;

        // To avoid deadlock when 3 sites want to insert on 3 edges, 
        // and they try to mark 3 pairs triangles: (a, b), (b, c), (c, a) 
        atomicMin(&ctags[pTri], x); 
        atomicMin(&ctags[decode_tri(ctriangles[pTri * 9 + (pOri + 1) % 3])], x); 
    }
}

/***************************************************************************
 * Determine which missing point insertion can be take place, 
 * mark those triangles that need to be deleted. 
 ***************************************************************************/
__global__ void kernelPreprocessTriangles(int *ctriangles, int *cvertarr, int *clocation, 
                                          int *ctags, int *tvertices, short *cnewtri, 
                                          int *cmarker, BYTE *caffected, int nVerts,
                                          int step, int *active, int noActive) {
    int t = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if ( t >= noActive )
        return ;

    int x = active[t]; 

    int pNextTri = clocation[x]; 

    if (pNextTri == MAXINT)
        return ; 

    int pTri = decode_tri(abs(pNextTri) - 1); 
    int pOri = decode_ori(abs(pNextTri) - 1); 
    int popp, pOppTri, pOppOri;

    bool success; 

    if (pNextTri >= 0)    // one triangle
        success = (ctags[pTri] == x); 
    else {
        popp = (ctriangles[pTri * 9 + (pOri + 1) % 3]); 
        pOppTri = decode_tri(popp); 
        pOppOri = decode_ori(popp); 
        success = (ctags[pTri] == x && ctags[pOppTri] == x); 
    }

    if (success) {
        cmarker[x] = 2;        
        cnewtri[pTri] = -step; 

        caffected[ctriangles[pTri * 9 + 3]] = 1; 
        caffected[ctriangles[pTri * 9 + 4]] = 1; 
        caffected[ctriangles[pTri * 9 + 5]] = 1; 

        if (pNextTri < 0) {
            cnewtri[pOppTri] = -step; 
            caffected[ctriangles[pOppTri * 9 + 3 + pOppOri]] = 1; 
        }
    } 
}

/************************************************************
 * Fix the vertex array for those affected sites 
 ************************************************************/
__global__ void kernelFixVertexArrayMissing(int *ctriangles, int *cvertarr, BYTE *caffected, 
                                            short *cnewtri, int nVerts) {    
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nVerts || caffected[x] != 1) 
        return ; 

    int p = cvertarr[x], pnext = p; 

    // Find the first valid triangle
    while (pnext >= 0 && cnewtri[decode_tri(pnext)] < 0)
        pnext = ctriangles[decode_tri(pnext) * 9 + 6 + decode_ori(pnext)]; 
    
    if (pnext != p)
        cvertarr[x] = pnext;

    while (pnext >= 0) {
        // Find an invalid triangle
        do {
            p = pnext; 
            pnext = ctriangles[decode_tri(p) * 9 + 6 + decode_ori(p)]; 
        } while (pnext >= 0 && cnewtri[decode_tri(pnext)] >= 0); 
        
        if (pnext >= 0)    {
            // Now pnext is deleted, so we fix the link for p. 

            // Find the next valid triangle
            while (pnext >= 0 && cnewtri[decode_tri(pnext)] < 0)
                pnext = ctriangles[decode_tri(pnext) * 9 + 6 + decode_ori(pnext)]; 
            
            ctriangles[decode_tri(p) * 9 + 6 + decode_ori(p)] = pnext; 
        }
    }
}

__global__ void kernelInsertMissingSites(int *ctriangles, int *cvertarr, int *clocation, 
                                         int *cmarker, int *cavailtri, int *cprefix, 
                                         short *cnewtri, int nVerts, int step,
                                         int *active, int noActive) 
{
    int t = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if ( t >= noActive )
        return;

    int x = active[t];

    if (cmarker[x] != 2)
        return ; 

    active[t] = -1; 

    int pNextTri = clocation[x]; 

    int pTri = decode_tri(abs(pNextTri) - 1); 
    int pOri = decode_ori(abs(pNextTri) - 1); 
    int pOrg, pDest, pApex, pOpposite; 
    int t1, t2, t3, t4, tmp; 

    int offset = cprefix[x]; 

    t1 = pTri;
    t2 = cavailtri[offset]; 
    t3 = cavailtri[offset + 1]; 

    pApex = ctriangles[pTri * 9 + 3 + pOri]; 
    pOrg = ctriangles[pTri * 9 + 3 + (pOri + 1) % 3];  
    pDest = ctriangles[pTri * 9 + 3 + (pOri + 2) % 3];  

    if (pNextTri >= 0) {    // one triangle
        INSERT_TRIANGLE(pOrg, pDest, x, t1); 
        INSERT_TRIANGLE(pDest, pApex, x, t2); 
        INSERT_TRIANGLE(pApex, pOrg, x, t3); 
    } else {
        int nDest = ctriangles[pTri * 9 + (pOri + 1) % 3]; 
        int pOppTri = decode_tri(nDest); 
        int pOppOri = decode_ori(nDest); 
        pOpposite = ctriangles[pOppTri * 9 + 3 + pOppOri]; 
        t4 = pOppTri; 

        INSERT_TRIANGLE(pOrg, pDest, x, t1); 
        INSERT_TRIANGLE(pDest, pOpposite, x, t2); 
        INSERT_TRIANGLE(pOpposite, pApex, x, t3); 
        INSERT_TRIANGLE(pApex, pOrg, x, t4); 
    }
}

/******************************************************************
 * Update the links between triangles after adding new triangles
 ******************************************************************/
__global__ void kernelUpdateMissingTriangleLinks(int *ctriangles, int *cvertarr, short *cnewtri, 
                                                 int nTris, int step) {
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris || cnewtri[x] != step) 
        return ; 

    int p0, p1, p2, n0 = -1, n1 = -1, n2 = -1;
    int nCounter, pNextTri, pTri, pOri, pTri9; 
    int x9 = x * 9;

    p2 = ctriangles[x9 + 3]; 
    p1 = ctriangles[x9 + 5]; 
    p0 = ctriangles[x9 + 4]; 
    nCounter = 0; 

    // orientation 0
    // Travel through the list of triangles sharing vertex 0 with this triangle.
    // In this list we can find at most two triangles sharing edge (p0, p1) and 
    // (p2, p0) with our triangle. 
    pNextTri = cvertarr[p0];

    while (pNextTri >= 0 && nCounter < 2) {
        pTri = decode_tri(pNextTri); 
        pOri = decode_ori(pNextTri); 
        pTri9 = pTri * 9; 

        if (p2 == ctriangles[pTri9 + 3 + (pOri + 2) % 3]) {    // NextDest
            n2 = pNextTri; 
            ctriangles[pTri9 + pOri] = (x << 2) | 2;  
            nCounter++; 
        }

        if (p1 == ctriangles[pTri9 + 3 + pOri]) {    // NextApex
            n0 = (pTri << 2) | ((pOri + 2) % 3);  
            ctriangles[pTri9 + (pOri + 2) % 3] = (x << 2);  
            nCounter++; 
        }

        pNextTri = ctriangles[pTri9 + 6 + pOri]; 
    }

    // orientation 1
    // Find the triangle with edge (p1, p2)
    pNextTri = cvertarr[p1]; 

    while (pNextTri >= 0) {
        pTri = decode_tri(pNextTri); 
        pOri = decode_ori(pNextTri); 
        pTri9 = pTri * 9; 

        if (p2 == ctriangles[pTri9 + 3 + pOri]) {    // NextApex
            n1 = (pTri << 2) | ((pOri + 2) % 3); 
            ctriangles[pTri9 + (pOri + 2) % 3] = (x << 2) | 1;  
            break ; 
        }

        pNextTri = ctriangles[pTri9 + 6 + pOri]; 
    }

    ctriangles[x9 + 0] = n0; 
    ctriangles[x9 + 1] = n1; 
    ctriangles[x9 + 2] = n2; 
}

/********************************************************************
 * Fix vertex array
 ********************************************************************/

__global__ void kernelMarkValidTriangles1(short *cnewtri, int *cvalid, int nTris)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris) 
        return ; 

    cvalid[x] = (cnewtri[x] >= 0) ? 1 : 0; 
}

__global__ void kernelCollectEmptySlots1(short *cnewtri, int *cprefix, int *cempty, int nTris)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris || cnewtri[x] >= 0) 
        return ; 

    int id = x - cprefix[x]; 

    cempty[id] = x; 
}

__global__ void kernelFillEmptySlots1(short *cnewtri, int *cprefix, int *cempty, int *ctriangles, 
                                     int nTris, int newnTris, int offset)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris || cnewtri[x] < 0) 
        return ; 

    int value;

    if (x < newnTris) 
        value = x; 
    else {
        value = cempty[cprefix[x] - offset]; 

        for (int i = 0; i < 9; i++)
            ctriangles[value * 9 + i] = ctriangles[x * 9 + i]; 
    }        

    cprefix[x] = value; 
}

__global__ void kernelFixIndices1(int *ctriangles, int *newindex, int nTris) {
    __shared__ int ct[WBLOCK * 9]; 

    int tId = threadIdx.x; 
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x, x9 = x * 9;
    int i, id; 
    
    if (x >= nTris)
        return ;

    // Cooperatively read all triangles processed by one block
    for (i = 0, id = tId; i < 9; i++, id += WBLOCK)
        ct[id] = ctriangles[x9 + id]; 

    __syncthreads(); 
    
    if (x + tId < nTris) {
        i = tId * 9;         
        if (ct[i] >= 0)
            ct[i] = encode_tri(newindex[decode_tri(ct[i])], decode_ori(ct[i])); 
        i++; 
        if (ct[i] >= 0)
            ct[i] = encode_tri(newindex[decode_tri(ct[i])], decode_ori(ct[i])); 
        i++; 
        if (ct[i] >= 0)
            ct[i] = encode_tri(newindex[decode_tri(ct[i])], decode_ori(ct[i])); 
        i++; 
        i++; 
        i++; 
        i++; 
        if (ct[i] >= 0)
            ct[i] = encode_tri(newindex[decode_tri(ct[i])], decode_ori(ct[i])); 
        i++; 
        if (ct[i] >= 0)
            ct[i] = encode_tri(newindex[decode_tri(ct[i])], decode_ori(ct[i])); 
        i++; 
        if (ct[i] >= 0)
            ct[i] = encode_tri(newindex[decode_tri(ct[i])], decode_ori(ct[i]));        
    }

    __syncthreads(); 

    for (i = 0, id = tId; i < 9; i++, id += WBLOCK)
        ctriangles[x9 + id] = ct[id]; 
}

/********************************************************************
 * Insert missing sites caused by overlapping or bad case shifting
 ********************************************************************/
void cudaMissing()
{
	hipFuncSetCacheConfig(reinterpret_cast<const void*>( kernelLocateTriangleContainer), hipFuncCachePreferL1 ); 
	hipFuncSetCacheConfig(reinterpret_cast<const void*>( kernelUpdateMissingTriangleLinks), hipFuncCachePreferL1 ); 

	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(constData), hostConst, 13 * sizeof(REAL)) ); 

    // Collect dead triangles, insert new triangles to these slots first. 
    int *cdeadtri, *cmarker, *cprefix; 

	cutilSafeCall( hipMalloc((void **) &cprefix, 2 * nVerts * sizeof(int)) ); 
	cutilSafeCall( hipMalloc((void **) &cmarker, 2 * nVerts * sizeof(int)) ); 

    dim3 block = dim3(WBLOCK); 
    dim3 grid = dim3(STRIPE, nTris / (STRIPE * block.x) + 1); 
    int lastItem; 

    // Mark all dead triangles as 1 in cmarker array, 0 if not. 
    kernelMarkDeadTriangles<<< grid, block >>>(cmarker, cnewtri, nTris); 
    cutilCheckError(); 

    cutilSafeCall( hipMemcpy(&lastItem, cmarker + nTris - 1, sizeof(int), hipMemcpyDeviceToHost) ); 

    // Use prefix sum to compute the offset
    thrust::exclusive_scan( IntDPtr(cmarker), IntDPtr(cmarker) + nTris, IntDPtr(cprefix) ); 

    // Temporary release what we no longer need. 
	cutilSafeCall( hipFree(cmarker));

    // Compute the size needed for the list of dead triangles
    // We also store the unused slots (after nTris but less than 2 * nVerts)
    int deadCounter; 
    cutilSafeCall( hipMemcpy(&deadCounter, cprefix + nTris - 1, sizeof(int), hipMemcpyDeviceToHost) ); 
    deadCounter += lastItem; 

    int tailTri = nVerts * 2 - nTris; 

    int deadListSize = deadCounter + tailTri; 

    cutilSafeCall( hipMalloc((void **) &cdeadtri, deadListSize * sizeof(int)) ); 

    // Collect these dead triangles into the 
    kernelCollectDeadTriangles<<< grid, block >>>(cdeadtri, cnewtri, cprefix, nTris); 
    cutilCheckError(); 

    //printf("Dead triangles: %i\n", deadCounter); 

     grid = dim3(256); //tailTri / block.x + 1);
    kernelFillIncrement<<< grid, block >>>(cdeadtri + deadCounter, nTris, tailTri); 
    cutilCheckError(); 
    /********************************************************
     * Process missing sites
     ********************************************************/
	int *active; 
    cutilSafeCall( hipMalloc( (void**)&active, nVerts * sizeof(int) ));
	IntDPtr activePtr( active );
    IntDPtr cvertarrPtr( cvertarr ); 

	thrust::sequence( activePtr, activePtr + nVerts );
    
	IntDPtr lastActivePtr = thrust::remove_if(
		activePtr, activePtr + nVerts, cvertarrPtr, isNotMinusOne() ); 

	int noActive = lastActivePtr - activePtr; 

    int *clocation;
    BYTE *caffected; 

	cutilSafeCall( hipMalloc((void **) &caffected, nVerts * sizeof(BYTE)) ); 
	cutilSafeCall( hipMalloc((void **) &clocation, nVerts * sizeof(int)) ); 
	cutilSafeCall( hipMalloc((void **) &cmarker, nVerts * sizeof(int)) ); 

    block = dim3(128); 
    dim3 gridFull = dim3(STRIPE, nVerts / (STRIPE * block.x) + 1);
    
    int triUsed = 0; 

    do {
        // cprefix will be used as a marker for voting which insertion can be processed
        cutilSafeCall( hipMemset(cprefix, 127, nVerts * 2 * sizeof(int)) ); 
        cutilSafeCall( hipMemset(cflag, 0, sizeof(int)) ); 

        // Locate triangles containing the missing sites
        grid = dim3(STRIPE, noActive / (STRIPE * block.x) + 1); 
        kernelLocateTriangleContainer<<< grid, block >>>(ctriangles, cvertarr,  
            tvertices, clocation, cprefix, covertices, nVerts, active, noActive); 
        cutilCheckError(); 

        cutilSafeCall( hipMemset(cmarker, 0, nVerts * sizeof(int)) ); 
        cutilSafeCall( hipMemset(caffected, 0, nVerts) ); 

        // Determine which missing point insertion can be done in this pass
        kernelPreprocessTriangles<<< grid, block >>>(ctriangles, cvertarr, clocation, 
            cprefix, tvertices, cnewtri, cmarker, caffected, nVerts, step,
            active, noActive); 

        // In cmarker we have the number of new triangles 
        // that will be generated by inserting each site (0 or 2). 
        thrust::exclusive_scan( IntDPtr(cmarker), IntDPtr(cmarker) + nVerts, IntDPtr(cprefix) ); 

        // We remove the container triangle and fix the vertex array. 
        kernelFixVertexArrayMissing<<< gridFull, block >>>(ctriangles, cvertarr, caffected, 
                                 cnewtri, nVerts);
        // We then insert three new triangles for each missing site inserted. 
        kernelInsertMissingSites<<< grid, block >>>(ctriangles, cvertarr, clocation, 
            cmarker, cdeadtri + triUsed, cprefix, cnewtri, nVerts, step, active, noActive); 

        // Update the offset in the dead triangle list
        cutilSafeCall( hipMemcpy(&lastItem, cmarker + nVerts - 1, sizeof(int), hipMemcpyDeviceToHost) ); 
    
        int used; 
        cutilSafeCall( hipMemcpy(&used, cprefix + nVerts - 1, sizeof(int), hipMemcpyDeviceToHost) ); 
        triUsed += used + lastItem; 
        
        int newsize = MAX(nTris, nTris - deadCounter + triUsed); 

        // Update links between the new triangles and the old one. 
        grid = dim3(STRIPE, newsize / (STRIPE * block.x) + 1); 
        kernelUpdateMissingTriangleLinks<<< grid, block >>>(ctriangles, cvertarr, cnewtri, newsize, step); 

        //printf("--------Insert missing sites - step %i ; Inserted %i triangles\n", step, used + lastItem);

        IntDPtr lastActivePtr = thrust::remove_if(
            activePtr, activePtr + noActive, isNegative() ); 

        noActive = lastActivePtr - activePtr; 
        
        step++; 

    } while (noActive > 0); 

    cutilSafeCall( hipFree( active ) ); 

    // We do not keep track of the dead triangles after this, 
    // because after removing the fake boundary, there would be a lot more, and they
    // will be recompute and recompact by then. 

    deadCounter -= triUsed; 

    // If we have used up all the dead triangles and more, we update nTris
    if (deadCounter < 0) 
        nTris -= deadCounter; 

    /******* DONE *******/   

	 /*********************************************************
     * Compact the triangle list 
    *********************************************************/

	cutilSafeCall( hipFree(cmarker));
	cutilSafeCall( hipFree(clocation));
	cutilSafeCall( hipFree(caffected));
	cutilSafeCall( hipFree(cprefix));
	cutilSafeCall( hipFree(tvertices));
	cutilSafeCall( hipFree(cdeadtri) ); 
	 /*********************************************************
     * Compact the triangle list 
     *********************************************************/
    if(deadCounter>0)
	{
		int *cvalid, *cprefix1;

		cutilSafeCall( hipMalloc((void **) &cvalid, 2 * nVerts * sizeof(int)) ); 
		cutilSafeCall( hipMalloc((void **) &cprefix1, 2 * nVerts * sizeof(int)) ); 

		block = dim3(WBLOCK); 
		grid = dim3(STRIPE, nTris / (STRIPE * block.x) + 1); 

		// Mark the valid triangles in the list
		kernelMarkValidTriangles1<<< grid, block >>>(cnewtri, cvalid, nTris); 
		cutilCheckError(); 

		// Compute the offset of them in the new list
        thrust::exclusive_scan( IntDPtr(cvalid), IntDPtr(cvalid) + nTris, IntDPtr(cprefix1) ); 

		int newnTris, lastitem, offset; 
		cutilSafeCall( hipMemcpy(&newnTris, cprefix1 + nTris - 1, sizeof(int), hipMemcpyDeviceToHost) ); 
		cutilSafeCall( hipMemcpy(&lastitem, cvalid + nTris - 1, sizeof(int), hipMemcpyDeviceToHost) ); 
		newnTris += lastitem; 
		cutilSafeCall( hipMemcpy(&offset, cprefix1 + newnTris, sizeof(int), hipMemcpyDeviceToHost) ); 

	//    printf("nTris = %i, new nTris = %i\n", nTris, newnTris); 

		// Find all empty slots in the list
		kernelCollectEmptySlots1<<< grid, block >>>(cnewtri, cprefix1, cvalid, nTris); 
		cutilCheckError(); 

		// Move those valid triangles at the end of the list
		// to the holes in the list. 
		grid = dim3(STRIPE, nTris / (STRIPE * block.x) + 1); 
		kernelFillEmptySlots1<<< grid, block >>>(cnewtri, cprefix1, cvalid, ctriangles, 
			nTris, newnTris, offset); 
		cutilCheckError(); 

		// Fix the links after the index of our triangles are mixed up
		grid = dim3(STRIPE, newnTris / (STRIPE * block.x) + 1); 
		kernelFixIndices1<<< grid, block >>>(ctriangles, cprefix1, newnTris); 
		cutilCheckError(); 


		cutilSafeCall( hipFree(cprefix1));
	    cutilSafeCall( hipFree(cvalid));
		nTris = newnTris; 
	}
}
