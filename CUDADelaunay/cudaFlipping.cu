#include "hip/hip_runtime.h"
/*
Author: Cao Thanh Tung, Qi Meng
Date: 15/03/2011

File Name: cudaFlipping.cu

This file include all CUDA code to perform the flipping step

===============================================================================

Copyright (c) 2011, School of Computing, National University of Singapore. 
All rights reserved.

Project homepage: http://www.comp.nus.edu.sg/~tants/cdt.html

If you use GPU-DT and you like it or have comments on its usefulness etc., we 
would love to hear from you at <tants@comp.nus.edu.sg>. You may share with us
your experience and any possibilities that we may improve the work/code.

===============================================================================

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution. 

Neither the name of the National University of University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission from the National University of Singapore. 

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES 
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.

*/

#pragma warning(disable: 4311 4312)

#include <hip/device_functions.h>
#include <stdio.h>
#include <string.h>
#include "gpudt.h"
#include "cudaDecl.h"
#include "common.h"

#include "cudaCCW.h"

/***********************************************************
* Declarations
***********************************************************/
#define WBLOCK      256        

#define SET_TRIANGLE(vOrg, vDest, vApex, nOrg, nDest, nApex, tri, ori) \
    ctriangles[(tri) * 9 + 3 + ((ori) + 1) % 3] = (vOrg); \
    ctriangles[(tri) * 9 + 3 + ((ori) + 2) % 3] = (vDest); \
    ctriangles[(tri) * 9 + 3 + (ori)] = (vApex); \
    ctriangles[(tri) * 9 + 6 + (ori)] = (nOrg); \
    ctriangles[(tri) * 9 + 6 + ((ori) + 1) % 3] = (nDest); \
    ctriangles[(tri) * 9 + 6 + ((ori) + 2) % 3] = (nApex) 

#define UPDATE_TEMP_LINK(pTriOri, pNext) \
    if ((pTriOri) >= 0) \
    ctriangles[decode_tri(pTriOri) * 9 + 6 + decode_ori(pTriOri)] = -(pNext)

#define UPDATE_LINK(pTriOri, pNext) \
    if ((pTriOri) >= 0) \
    ctriangles[decode_tri(pTriOri) * 9 + decode_ori(pTriOri)] = (pNext)

/**************************************************************
* Exported methods
**************************************************************/
extern "C" int cudaFlipping(int **suspective); 

/**************************************************************
* Definitions
**************************************************************/
// Decode an oriented triangle. 
// An oriented triangle consists of 32 bits. 
// - 30 highest bits represent the triangle index, 
// - 2 lowest bits represent the orientation (the starting vertex, 0, 1 or 2)
#define decode_tri(x)            ((x) >> 2)
#define decode_ori(x)            ((x) & 3)
#define encode_tri(tri, ori)     (((tri) << 2) | (ori))

/************************************************************
* Variables and functions shared with the main module
************************************************************/
extern int nTris, nVerts, nPoints, nConstraints;   
extern int *ctriangles;            
extern REAL2 *covertices;        
extern int *cflag; 
extern int *cconstraints;
extern int *cvertarr;
extern BYTE *ifEdgeIsConstraint_cpu;
extern PGPUDTPARAMS  gpudtParams;

/***************************************************************************
* Determine which edge is constraint edge, 
* mark those edges. 
***************************************************************************/
__global__ void KernelMarkConstrain1(int *ctriangles, int *cvertarr, REAL2 *covertices, 
                                     int* cflag, int nConstraints, int* tconstrainLink_flip, BYTE *ifEdgeIsConstraint)
{

    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x+threadIdx.x;
    if (x >= nConstraints) 
        return ;

    int point1 = tconstrainLink_flip[2*x];
    int point2 = tconstrainLink_flip[2*x+1];	
    REAL2 point1_R,point2_R,point3_R;
    point1_R = covertices[point1];
    point2_R = covertices[point2];
    REAL test1=-1;
    int ApTri, ApOri, adjacent;
    int pTri, pOri, pApex;	
    int p = cvertarr[point1]; 	

    pTri = decode_tri(p);   pOri = decode_ori(p);
    pApex = ctriangles[pTri*9 + 3 +  pOri];	
    point3_R = covertices[pApex];
    test1 = cuda_ccw(point1_R, point2_R, point3_R);
    int pnext = -1;
    if(test1==0 && point2==pApex)//point2 == pApex
    {		

        ifEdgeIsConstraint[pTri*3 + (pOri+2)%3] = 1; 
        adjacent = ctriangles[pTri*9 + (pOri+2)%3];
        ApTri  = decode_tri(adjacent);
        if(ApTri>=0)
        {
            ApOri  = decode_ori(adjacent);
            ifEdgeIsConstraint[ApTri*3 + (ApOri+0)%3] = 1; 
        }
        return;
    }
    else if(test1<0)//pApex is on the right
    {

        pnext = ctriangles[pTri*9 + (pOri+2)%3];		
        do{
            p = pnext;
            pTri = decode_tri(p);   pOri = decode_ori(p);
            pApex = ctriangles[pTri*9 + 3 +  pOri];				
            if(pApex == point2)
            {
                ifEdgeIsConstraint[pTri*3 + (pOri+2)%3] = 1; 
                adjacent = ctriangles[pTri*9 + (pOri+2)%3];
                ApTri  = decode_tri(adjacent);				

                if(ApTri>=0)
                {					  
                    ApOri  = decode_ori(adjacent);
                    ifEdgeIsConstraint[ApTri*3 + (ApOri+0)%3] = 1;					
                }
                return;
            }
            else
            {
                pnext = ctriangles[pTri*9 + (pOri+2)%3];	
            }

        }while(true);
    }
    else//pApex is on the left  or test==0&&pApex!=point2		
    {
        if( ctriangles[pTri*9 + 3 +  (pOri+2)%3] == point2)//the first triangle is the needed triangle
        {
            ifEdgeIsConstraint[pTri*3 + (pOri+0)%3] = 1;
            adjacent = ctriangles[pTri*9 + (pOri+0)%3];
            ApTri  = decode_tri(adjacent);
            if(ApTri>=0)
            {
                ApOri  = decode_ori(adjacent);
                ifEdgeIsConstraint[ApTri*3 + (ApOri+0)%3] = 1; 
            }
            return;
        } 

        pnext = ctriangles[pTri*9 + (pOri+0)%3];	

        do{

            p = pnext;			
            if(decode_tri(p)<0)
            {
                ifEdgeIsConstraint[pTri*3 + (pOri+0)%3] = 1;
                return;
            }
            pTri = decode_tri(p);   pOri = decode_ori(p);			
            pApex = ctriangles[pTri*9 + 3 +  (pOri+0)%3];//pApex
            if(pApex == point2)
            {				
                ifEdgeIsConstraint[pTri*3 + (pOri+1)%3] = 1;
                adjacent = ctriangles[pTri*9 + (pOri+1)%3];				
                ApTri  = decode_tri(adjacent);
                if(ApTri>=0)
                {
                    ApOri = decode_ori(adjacent);
                    ifEdgeIsConstraint[ApTri*3 + (ApOri+0)%3] = 1; 
                }
                return;
            }
            else
            {
                pnext = ctriangles[pTri*9 + (pOri+1)%3];	
            }
        }while(true);
    } 
}

/*************************************************************
* Detect all sites that can be shifted together 
* without causing any crossing. 
* We're guaranteed that all boundary sites are already marked
*************************************************************/
__global__ void kernelNeedFlipping(int *ctriangles, REAL2 *covertices, BYTE *flipOri, 
                                   int *flipStep, int *flipBy, BYTE *cmarker, 
                                   int nTris, int step, BYTE *ifEdgeIsConstrain,
                                   int *active, int noActive) 
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if ( x >= noActive )
        return ;

    x = active[x];

    if ( cmarker[x] < 0 )
        return ; 
    
	int x9 = x * 9;
    int i, id; 

    REAL2 pOrg, pDest, pApex, pOpp;
    int nOrg, nDest, nApex; 
    int pTri, pOri; 
    bool notsure = false; 

    nOrg = ctriangles[x9 + 0]; 
    nDest = ctriangles[x9 + 1]; 
    nApex = ctriangles[x9 + 2];

    pOrg = covertices[ctriangles[x9 + 4]]; 
    pDest = covertices[ctriangles[x9 + 5]]; 
    pApex = covertices[ctriangles[x9 + 3]];

    // nOrg
    int t;
    t = ifEdgeIsConstrain[x * 3 + 0];
    if( t == -1)// do when current edge is not constraint edge
    {
        pTri = decode_tri(nOrg); pOri = decode_ori(nOrg); 
        if (x < pTri || (pTri >= 0 && cmarker[pTri] != 0)) {
            pOpp = covertices[ctriangles[pTri * 9 + 3 + pOri]]; 

            switch (cuda_inCircle(pOrg, pDest, pApex, pOpp)) {
                case 0 : notsure = true; break;     
                case 1 : 
                    flipOri[x] = 2; 
                    flipStep[x] = step; 
                    atomicMin(&flipBy[x], x); 
                    atomicMin(&flipBy[pTri], x);
                    return ; 
            }
        }
    }

    // nDest
    t = ifEdgeIsConstrain[x * 3 + 1];
    if( t == -1) //do when current edge is not constraint edge
    {
        pTri = decode_tri(nDest); pOri = decode_ori(nDest); 
        if (x < pTri || (pTri >= 0 && cmarker[pTri] != 0)) {
            pOpp = covertices[ctriangles[pTri * 9 + 3 + pOri]]; 

            switch (cuda_inCircle(pDest, pApex, pOrg, pOpp)) {
                case 0 : notsure = true; break; 
                case 1 : 
                    flipOri[x] = 0; 
                    flipStep[x] = step; 
                    atomicMin(&flipBy[x], x); 
                    atomicMin(&flipBy[pTri], x);
                    return ; 
            }
        }
    }

    // nApex
    pTri = decode_tri(nApex); pOri = decode_ori(nApex); 
    t = ifEdgeIsConstrain[x * 3 + 2];
    if( t == -1)//do when current edge is not constraint edge
    {
        if (x < pTri || (pTri >= 0 && cmarker[pTri] != 0)) {
            pOpp = covertices[ctriangles[pTri * 9 + 3 + pOri]]; 

            switch (cuda_inCircle(pApex, pOrg, pDest, pOpp)) {
                case 0 : notsure = true; break; 
                case 1 : 
                    flipOri[x] = 1; 
                    flipStep[x] = step; 
                    atomicMin(&flipBy[x], x);       // The one with minimun index
                    atomicMin(&flipBy[pTri], x);    // will win. 
                    return ; 
            }
        }
    }

    if (notsure)    // Inaccurate in_circle test.
        flipStep[x] = -step; 
	
	cmarker[x] = -1;
}

// To flip a pair of triangle, we need to update the links between neighbor
// triangles. There is a chance that we need to flip two pairs in which 
// there are two neighboring triangles, thus the flipping is performed
// in 3 phases: 
// - Phase 1: Update the triangle vertices. Also, note down the new 
//            neighbors of each triangles in the 3 nexttri links (previously
//            used to store the vertex array). 
// - Phase 2: Each new triangle will then inform its new neighbors of its 
//            existence by writing itself in the corresponding nexttri link
//            of its neighbor.
// - Phase 3: Each new triangle update its links to its neighbors using the
//            3 new nexttri links. If any of the link is not updated, that
//            means the corresponding neighbor is not flipped in this pass. 
//            We then actively update the corresponding link in that triangle.
__global__ void kernelUpdatePhase1(int *ctriangles, BYTE *cflipOri, int *cflipStep, 
                                   int *cflipBy, BYTE *cmarker, int nTris, int step,
                                   BYTE *ifEdgeIsConstraint, int *active, int noActive) 
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= noActive)
        return ;

    x = active[x];

    if ( cmarker[x] < 0 )
        return; 

    if (cflipStep[x] < 0) {  // Inaccurate incircle test
        return ; 
    }

    int ori, pOpp, pOppTri, pOppOri, pOrg, pDest, pApex, nOrg, nApex, npDest, npApex;
    int newtag = -1; 

    if (cflipBy[x] == x) {      // I'm the one who win the right to flip myself
        ori = cflipOri[x]; 
        pOpp = ctriangles[x * 9 + (ori + 1) % 3]; 
        pOppTri = decode_tri(pOpp);
        pOppOri = decode_ori(pOpp); 

        if (cflipBy[pOppTri] == x) {    // I'm also the one who win the right
            newtag = pOpp;              // to flip this neighbor, so I can flip.
            pOrg = ctriangles[x * 9 + 3 + (ori + 1) % 3]; 
            pDest = ctriangles[x * 9 + 3 + (ori + 2) % 3]; 
            pApex = ctriangles[x * 9 + 3 + ori];
            pOpp = ctriangles[pOppTri * 9 + 3 + pOppOri]; 
            nOrg = ctriangles[x * 9 + ori]; 
            nApex = ctriangles[x * 9 + (ori + 2) % 3]; 
            npDest = ctriangles[pOppTri * 9 + (pOppOri + 1) % 3]; 
            npApex = ctriangles[pOppTri * 9 + (pOppOri + 2) % 3]; 

            // Update vertices + nexttri links
            SET_TRIANGLE(pOrg, pDest, pOpp, 3 + nOrg, -1, 3 + nApex, x, ori); 
            SET_TRIANGLE(pApex, pOrg, pOpp, -1, 3 + npDest, 3 + npApex, pOppTri, pOppOri); 
            int temp2,tempp1;

            temp2 = ifEdgeIsConstraint[x*3+(ori+2)%3];///old dest
            ifEdgeIsConstraint[x*3+(ori+2)%3]= -1;//dest

            tempp1 = ifEdgeIsConstraint[pOppTri*3+(pOppOri+1)%3];   //old org
            ifEdgeIsConstraint[pOppTri*3+(pOppOri+1)%3]= -1;//org

			ifEdgeIsConstraint[x*3+(ori+1)%3]= tempp1;//org
            ifEdgeIsConstraint[pOppTri*3+pOppOri]= temp2;//apex
       }
    }

    // Record the opp triangle to be used in the next phase
    cflipStep[x] = newtag;  
}

__global__ void kernelUpdatePhase2(int *ctriangles, BYTE *cflipOri, int *cflipStep, 
                                   int *cflipBy, BYTE *cmarker, int nTris, int step,
                                   int *active, int noActive) 
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= noActive )
        return;

    x = active[x];

    if ( cmarker[x] < 0 )
        return ; 
    
    if ( cflipStep[x] < 0) 
        return ; 

    int ori, pOpp, pOppTri, pOppOri, nOrg, nApex, npDest, npApex;

    ori = cflipOri[x]; 
    pOpp = cflipStep[x]; 
    pOppTri = decode_tri(pOpp);
    pOppOri = decode_ori(pOpp); 
    nOrg = ctriangles[x * 9 + ori]; 
    nApex = ctriangles[x * 9 + (ori + 2) % 3]; 
    npDest = ctriangles[pOppTri * 9 + (pOppOri + 1) % 3]; 
    npApex = ctriangles[pOppTri * 9 + (pOppOri + 2) % 3]; 

    // Update my neighbors of my existence
    UPDATE_TEMP_LINK(nOrg, encode_tri(x, ori)); 
    UPDATE_TEMP_LINK(nApex, encode_tri(pOppTri, pOppOri)); 
    UPDATE_TEMP_LINK(npDest, encode_tri(x, (ori + 1) % 3)); 
    UPDATE_TEMP_LINK(npApex, encode_tri(pOppTri, (pOppOri + 2) % 3)); 
}

__global__ void kernelUpdatePhase3(int *ctriangles, BYTE *cflipOri, int *cflipStep, 
                                   int *cflipBy, BYTE *cmarker, int nTris, int step,
                                   int *active, int noActive, bool collectMode) 
{
    int t = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if ( t >= noActive )
        return ;

    int x = active[t];

    if ( cmarker[x] < 0 )
    {
        if (collectMode)
        {
            active[t] = -1;
            active[t + noActive] = -1;
        }
        
        return; 
    } 

    if (cflipStep[x] < 0)
    {
        if ( collectMode )
        {
            active[t + noActive] = -1;
        }
        
        return ;
    }

    int ori, pOpp, pOppTri, pOppOri, nOrg, nApex, npDest, npApex;

    ori = cflipOri[x]; 
    pOpp = cflipStep[x]; 
    pOppTri = decode_tri(pOpp);
    pOppOri = decode_ori(pOpp); 

    // Update other links
    nOrg = ctriangles[x * 9 + 6 + ori]; 
    nApex = ctriangles[x * 9 + 6 + (ori + 2) % 3]; 
    npDest = ctriangles[pOppTri * 9 + 6 + (pOppOri + 1) % 3]; 
    npApex = ctriangles[pOppTri * 9 + 6 + (pOppOri + 2) % 3]; 

    if (nOrg > 0) {        // My neighbor do not update me, update him
        nOrg = -(nOrg - 3); 
        UPDATE_LINK(-nOrg, encode_tri(x, ori)); 
    }

    if (nApex > 0) {
        nApex = -(nApex - 3); 
        UPDATE_LINK(-nApex, encode_tri(pOppTri, pOppOri)); 
    }

    if (npDest > 0) {
        npDest = -(npDest - 3); 
        UPDATE_LINK(-npDest, encode_tri(x, (ori + 1) % 3)); 
    }

    if (npApex > 0) {
        npApex = -(npApex - 3); 
        UPDATE_LINK(-npApex, encode_tri(pOppTri, (pOppOri + 2) % 3)); 
    }

    // Update my own links
    ctriangles[x * 9 + ori] = -nOrg; 
    ctriangles[x * 9 + (ori + 1) % 3] = -npDest; 
    ctriangles[x * 9 + (ori + 2) % 3] = encode_tri(pOppTri, (pOppOri + 1) % 3); 
    ctriangles[pOppTri * 9 + pOppOri] = -nApex; 
    ctriangles[pOppTri * 9 + (pOppOri + 1) % 3] = encode_tri(x, (ori + 2) % 3); 
    ctriangles[pOppTri * 9 + (pOppOri + 2) % 3] = -npApex; 

    // Mark the affected triangles, so that we will check again in the next pass
    cflipStep[x] = -1; 

    if ( collectMode )
    {
        if ( cmarker[pOppTri] < 0 )
            active[t + noActive] = pOppTri;
        else
            active[t + noActive] = -1; 
    }
    else
    {
        cmarker[pOppTri] = 0;
    }

//    if (nOrg <= 0) cmarker[decode_tri(-nOrg)] = 0; 
//    if (nApex <= 0) cmarker[decode_tri(-nApex)] = 0; 
//    if (npDest <= 0) cmarker[decode_tri(-npDest)] = 0; 
//    if (npApex <= 0) cmarker[decode_tri(-npApex)] = 0; 
}

__global__ void kernelClearMarker( int *active, int noActive, BYTE *cmarker )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if ( x >= noActive )
        return ;

    cmarker[ active[x] ] = 0; 
} 

int cudaFlipping(int **suspective) 
{
    // Constants for the EXACT tests
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(constData), hostConst, 13 * sizeof(REAL)) ); 

    int step = 2; 

    BYTE *ifEdgeIsConstraint;

    hipMalloc( (void**)&ifEdgeIsConstraint, 3*nTris* sizeof(BYTE) );
    cutilSafeCall( hipMemset(ifEdgeIsConstraint, -1, 3*nTris * sizeof(BYTE)) );	

    int *cflipStep, *cflipBy;
    BYTE *cmarker, *cflipOri; 
    dim3 grid, block; 

    cutilSafeCall( hipMalloc((void **) &cflipOri, nVerts * 2) ); 
    cutilSafeCall( hipMalloc((void **) &cflipStep, nVerts * 2 * sizeof(int)) ); 
    cutilSafeCall( hipMalloc((void **) &cflipBy, nVerts * 2 * sizeof(int)) ); 
    cutilSafeCall( hipMalloc((void **) &cmarker, nVerts * 2) ); 

    cutilSafeCall( hipMemset(cflipStep, -1, nTris * sizeof(int)) ); 
    cutilSafeCall( hipMemset(cmarker, 0, nTris) ); 

	int *active; 
    cutilSafeCall( hipMalloc( (void**)&active, nTris * sizeof(int) ));
	IntDPtr activePtr( active ), lastActivePtr;
    ByteDPtr markerPtr( cmarker ); 

	thrust::sequence( activePtr, activePtr + nTris );
    
    int noActive = nTris;  

    block = dim3(128); 
    grid = dim3(STRIPE, nTris / (STRIPE * block.x) + 1); 
    // For all traingles, mark all constraint edge
    KernelMarkConstrain1<<<grid,block>>>(ctriangles, cvertarr, covertices, cflag, nConstraints, cconstraints,ifEdgeIsConstraint); 	
    cutilCheckError();

    bool collectMode = false;
    int count = 0; 

    // Start flipping
    do {
        cutilSafeCall( hipMemset(cflag, 0, sizeof(int)) ); 
        cutilSafeCall( hipMemset(cflipBy, 127, nTris * sizeof(int)) ); 

        // Perform incircle tests
        block = dim3(WBLOCK); 
        grid = dim3(STRIPE, noActive / (STRIPE * block.x) + 1); 
        kernelNeedFlipping<<< grid, block >>>(ctriangles, covertices, cflipOri, cflipStep, 
            cflipBy, cmarker, nTris, step, ifEdgeIsConstraint, active, noActive); 
        cutilCheckError(); 

        kernelUpdatePhase1<<< grid, block >>>(ctriangles, cflipOri, cflipStep, 
            cflipBy, cmarker, nTris, step, ifEdgeIsConstraint, active, noActive);	
        cutilCheckError();         

        kernelUpdatePhase2<<< grid, block >>>(ctriangles, cflipOri, cflipStep, 
            cflipBy, cmarker, nTris, step, active, noActive); 
        cutilCheckError(); 

        kernelUpdatePhase3<<< grid, block >>>(ctriangles, cflipOri, cflipStep, 
            cflipBy, cmarker, nTris, step, active, noActive, collectMode); 
        cutilCheckError();

        if ( collectMode )
        {
            lastActivePtr = thrust::remove_if(
                activePtr, activePtr + (noActive * 2), isNegative() );
            noActive = lastActivePtr - activePtr;

            grid = dim3( noActive / block.x + 1 );
            kernelClearMarker<<< grid, block >>>( active, noActive, cmarker ); 

            if ( noActive >= nTris / 2 )
                collectMode = false; 
        }
        else
        {
            count++; 

            if ( count % 5 == 0 )
            {
                thrust::sequence( activePtr, activePtr + nTris ); 
                lastActivePtr = thrust::remove_if(
                    activePtr, activePtr + nTris, markerPtr, isNegative() );
                noActive = lastActivePtr - activePtr;

                if ( noActive < nTris / 2 && noActive < WBLOCK * STRIPE )
                    collectMode = true;

                count = 0; 
            }
        }

        //printf( "%s, count = %i, noActive = %i\n",
        //    collectMode ? "collect" : "compact", count, noActive ); 
                
        step++; 

    } while (count > 0 || noActive > 0);



    // All the suspectice incircle tests will be noted down 
    // and let the CPU perform the exact incircle tests.
    int marker = step - 1; 

    *suspective = (int *) malloc(nTris * sizeof(int)); 
    cutilSafeCall( hipMemcpy(*suspective, cflipStep, nTris * sizeof(int), hipMemcpyDeviceToHost) ); 

    ifEdgeIsConstraint_cpu =  new BYTE[3*nTris];
    cutilSafeCall( hipMemcpy(ifEdgeIsConstraint_cpu, ifEdgeIsConstraint, 3*nTris * sizeof(BYTE), hipMemcpyDeviceToHost) ); 

    cutilSafeCall( hipFree(cmarker) ); 
    cutilSafeCall( hipFree(active) ); 
    cutilSafeCall( hipFree(cflipBy) ); 
    cutilSafeCall( hipFree(cflipStep) ); 
    cutilSafeCall( hipFree(cflipOri) ); 	
    cutilSafeCall( hipFree(ifEdgeIsConstraint) );
    cutilSafeCall( hipFree(cconstraints) );
    cutilSafeCall( hipFree(cvertarr) );

    return marker; 	
}

